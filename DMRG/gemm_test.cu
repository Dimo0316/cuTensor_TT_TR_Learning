#include<iostream>
#include<fstream>
#include<hip/hip_runtime.h>
#include<hipblas.h>
#include<hipsolver.h>
#include<hiprand.h>
#include<cmath>
#include <hiptensor.h>
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <string>
#include <unordered_map>
#include <vector>
#include<ctime>

using namespace std;
typedef float dt;


void permute(dt *d_L,int a,int b,int c,int d,int e,hiptensorHandle_t tensor_handle,int flag)
{	
	hipDataType typeA = HIP_R_32F;
    hipDataType typeC = HIP_R_32F;
    hipDataType typeCompute = HIP_R_32F;

    std::vector<int> modeA{'a','b','c','d','e'};
    std::vector<int> modeC(5);
    if(flag == 0)
    {    	
    	//std::vector<int> modeC{'e','a','d','b','c'};
    	modeC[0]='e';modeC[1]='a';modeC[2]='d';modeC[3]='b';modeC[4]='c';
    }
    else
    {   	
    	//std::vector<int> modeC{'a','d','e','b','c'};
    	modeC[0]='a';modeC[1]='d';modeC[2]='e';modeC[3]='b';modeC[4]='c';
    }

    int nmodeA = modeA.size();
    int nmodeC = modeC.size();
    std::unordered_map<int, int64_t> extent;

    extent['a'] = a;
    extent['b'] = b;
    extent['c'] = c;
    extent['d'] = d;
    extent['e'] = e;

    std::vector<int64_t> extentA;
    for (auto mode : modeA)
        extentA.push_back(extent[mode]);
    std::vector<int64_t> extentC;
    for (auto mode : modeC)
        extentC.push_back(extent[mode]);

    size_t elementsA = 1;
    for (auto mode : modeA)
        elementsA *= extent[mode];
    size_t elementsC = 1;
    for (auto mode : modeC)
        elementsC *= extent[mode];

    //size_t sizeA = sizeof(float) * elementsA;
    size_t sizeC = sizeof(float) * elementsC;

    dt *d_C;
    hipMalloc((void**)&d_C,sizeC);

    hiptensorHandle_t handle;
    cutensorInit(&handle);

    hiptensorTensorDescriptor_t descA;
    hiptensorInitTensorDescriptor(&handle,
                 &descA,
                 nmodeA,
                 extentA.data(),
                 NULL /* stride */,
                 typeA, HIPTENSOR_OP_IDENTITY);
    hiptensorTensorDescriptor_t descC;
    hiptensorInitTensorDescriptor(&handle,
                 &descC,
                 nmodeC,
                 extentC.data(),
                 NULL /* stride */,
                 typeC, HIPTENSOR_OP_IDENTITY);
    const float one = 1.0f;
    hiptensorPermutation(&handle,
                 &one, d_L, &descA, modeA.data(),
                       d_C, &descC, modeC.data(),
                 typeCompute, 0 /* stream */);
    hipDeviceSynchronize();

    d_L = d_C;
    if (d_C) hipFree(d_C);
}
void ncon(dt *d_L,dt *d_M,dt *d_A,int n,dt *d_out,hipblasHandle_t handle,hiptensorHandle_t tensor_handle)
{	
	dt *d_LM;
	hipMalloc((void**)&d_LM,sizeof(dt)*n *n*n*n*n);

	dt alpha = 1.0;
	dt beta = 0.0;
	hipblasSgemm(handle,
	            HIPBLAS_OP_N,HIPBLAS_OP_T,
	            n* n,n*n*n,n,
	            &alpha,d_L,n*n,d_M,n*n*n,
	            &beta,d_LM,n* n
	            );
	hipDeviceSynchronize();

	//[2,4,-2][1,2,4,5,-3]  
	// d_LM permute ->[-3,1,5,2,4]
	// (A+)(LM)
	permute(d_LM,n,n,n,n,n,tensor_handle,0);
	dt *d_ALM;
	hipMalloc((void**)&d_ALM,sizeof(dt)*n*n*n*n);
	hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_T,
	            n,n*n*n,n*n,
	            &alpha,d_A,n*n,d_LM,n*n*n,
	            &beta,d_ALM,n
	            );
	// A * ALM 
	//[1,5,-1][-2,-3,1,5]
  	hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_T,
  	            n,n*n,n*n,
  	            &alpha,d_A,n*n,d_ALM,n*n,
  	            &beta,d_out,n
  	            );
  	
  	hipDeviceSynchronize();

  	hipFree(d_LM);
  	hipFree(d_ALM);
}
int main()
{
	int n = 5;

    float *A = new float[n*n*n];
    float *B = new float[n*n*n*n];
    float *C = new float[n*n*n];
    float *D = new float[n*n*n]; // C=D

    for (int i = 0; i < n*n*n; i++)
        A[i] = (((float) rand())/RAND_MAX - 0.5)*100;
    for (int i = 0; i < n*n*n*n; i++)
        B[i] = (((float) rand())/RAND_MAX - 0.5)*100;
    for (int i = 0; i < n*n*n; i++)
        C[i] = (((float) rand())/RAND_MAX - 0.5)*100;
    for (int i = 0; i < n*n*n; i++)
       D[i]=C[i];

    float *A_d,*B_d,*C_d;
    hipMalloc((void**)&A_d,sizeof(float)*n*n*n);
    hipMalloc((void**)&B_d,sizeof(float)*n*n*n*n);
    hipMalloc((void**)&C_d,sizeof(float)*n*n*n);
    
    hipMemcpy(A_d,A,sizeof(float)*n*n*n,hipMemcpyHostToDevice);
    hipMemcpy(B_d,B,sizeof(float)*n*n*n,hipMemcpyHostToDevice);
    hipMemcpy(C_d,C,sizeof(float)*n*n*n,hipMemcpyHostToDevice);

    float *out_d;
    hipMalloc((void**)&out_d,sizeof(float)*n*n*n);

	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hiptensorHandle_t tensor_handle;
    cutensorInit(&tensor_handle); 

     float time_elapsed=0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
	hipEventRecord( start,0);

    ncon(A_d,B_d,C_d,n,out_d,handle,tensor_handle);
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_elapsed, start, stop);

    cout<<"cost time is :"<<time_elapsed<<endl;





}