#include<iostream>
#include<fstream>
#include<hip/hip_runtime.h>
#include<hipblas.h>
#include<hipsolver.h>
#include<hiprand/hiprand.h>
#include<cmath>
using namespace std;
typedef float dt;

__global__  void floattohalf(dt *AA,half *BB,long m){
	long i = blockIdx.x*blockDim.x+threadIdx.x;
	const long temp = blockDim.x*gridDim.x;
	if(i<m){
		BB[i]=__float2half(AA[i]);
		i+=temp;
	}
	__syncthreads();
}

void f2h(dt *A,half *B,long num){
	dim3 threads(512,1,1);
	dim3 blocks((num+512-1)/512,1,1);	
	floattohalf<<<blocks,threads>>>(A,B,num);
}

int main()
{
	for(int hh = 100;hh<160;hh=hh+160)
	{
	dt *A,*B,*C;
	int a=hh;
	int r=hh;
	int b=hh;
	hipHostAlloc((void**)&A,sizeof(dt)*a*r,0);
	hipHostAlloc((void**)&B,sizeof(dt)*r*b,0);
	hipHostAlloc((void**)&C,sizeof(dt)*b*a,0);
	srand(5);
	for(long long i = 0;i<a*r;i++){
		A[i] = rand()*0.1/(RAND_MAX*0.1);		
	}	
	for(long long i = 0;i<r*b;i++){
		B[i] = rand()*0.1/(RAND_MAX*0.1);		
	}

	dt alpha = 1.0;
	dt beta = 0.0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	//cublasMath_t CUBLAS_TENSOR_OP_MATH;
	hipblasSetMathMode(handle,HIPBLAS_TENSOR_OP_MATH);

	dt *d_A;
	dt *d_B;
	dt *d_C;

		
		half *h_A,*h_B;
		hipMalloc((void **)&h_A,sizeof(half)*a*r);
		hipMalloc((void **)&h_B,sizeof(half)*b*r);

	hipMalloc((void **)&d_A,sizeof(dt)*a*r);
	hipMalloc((void **)&d_B,sizeof(dt)*r*b);
	hipMalloc((void **)&d_C,sizeof(dt)*a*b);
	hipMemcpyAsync(d_A,A,sizeof(dt)*a*r,hipMemcpyHostToDevice,0);
	hipMemcpyAsync(d_B,B,sizeof(dt)*r*b,hipMemcpyHostToDevice,0);
	hipDeviceSynchronize();

		f2h(d_A,h_A,a*r);
		f2h(d_B,h_B,b*r);
		hipDeviceSynchronize();

	hipblasGemmEx(handle,HIPBLAS_OP_N,
		             HIPBLAS_OP_N,
		             a,
		             b,
		             r,
		             &alpha,
		             h_A,
		             HIP_R_16F,
		             a,
		             h_B,
		             HIP_R_16F,
		             r,
		             &beta,
		             d_C,
		             HIP_R_32F,
		             a,
		             HIP_R_32F,
		             CUBLAS_GEMM_DEFAULT_TENSOR_OP);
	hipDeviceSynchronize();

	hipMemcpy(C,d_C,sizeof(dt)*a*b,hipMemcpyDeviceToHost);

	for(int i=0;i<10;i++)
	{
		cout<<C[i]<<" _ ";
	}
	cout<<endl;
}
}

