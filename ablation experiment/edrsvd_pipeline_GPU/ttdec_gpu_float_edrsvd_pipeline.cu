#include "hip/hip_runtime.h"
#include "head.h"

hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
hipblasGemmAlgo_t algo = HIPBLAS_GEMM_DEFAULT; //CUBLAS_GEMM_DEFAULT_TENSOR_OP HIPBLAS_GEMM_DEFAULT
hipblasMath_t mathMode = HIPBLAS_DEFAULT_MATH; //HIPBLAS_TENSOR_OP_MATH  HIPBLAS_DEFAULT_MATH
hipError_t cudaStat1 = hipSuccess;
hipError_t cudaStat2 = hipSuccess;
hipError_t cudaStat3 = hipSuccess;
hipError_t cudaStat4 = hipSuccess;
hipError_t cudaStat5 = hipSuccess;
hipError_t cudaStat6 = hipSuccess;
hipError_t cudaStat7 = hipSuccess;
bool CalculateError = true;
const float alpha = 1.0, beta = 0.0;
const double tol = 1.e-12;

double norm2HH(float *A, long long len){
  double norm2Result = 0.0;
  for(long long i = 0; i < len; ++i){
    norm2Result += (A[i] * A[i]);
  }
  norm2Result = sqrtf(norm2Result);
  return norm2Result;
}

void basicEig(hipsolverHandle_t cusolverH, hipblasHandle_t cublasH, dt *d_A, const int m, dt *d_W){  
    int *devInfo = NULL;
    dt *d_work = NULL;
    int  lwork = 0;
    int info_gpu = 0;
    // printf("eign %d\n", m);
    hipMalloc ((void**)&devInfo, sizeof(int));
    
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    cusolver_status = hipsolverDnSsyevd_bufferSize(
        cusolverH,
        jobz,
        uplo,
        m,
        d_A,
        m,
        d_W,
        &lwork);
    assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS);

    cudaStat1 = hipMalloc((void**)&d_work, sizeof(dt)*lwork);
    assert(hipSuccess == cudaStat1);

// step 4: compute spectrum
    cusolver_status = hipsolverDnSsyevd(
        cusolverH,
        jobz,
        uplo,
        m,
        d_A,
        m,
        d_W,
        d_work,
        lwork,
        devInfo);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    assert(hipSuccess == cudaStat1);

    cudaStat1 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);

    // printf("after syevd: info_gpu = %d\n", info_gpu);
    assert(0 == info_gpu);

    if (devInfo) hipFree(devInfo);
    if (d_work ) hipFree(d_work);
    // if (d_W    ) hipFree(d_W);
}

void basicQR(hipsolverHandle_t cusolverH, float *d_A,int m,int n)
{
    float *d_work = NULL, *d_tau = NULL;
    int *devInfo = NULL;
     int lwork_geqrf = 0;
    int lwork_orgqr = 0;
    int lwork = 0;
    int info_gpu = 0;
    hipMalloc((void**)&d_tau, sizeof(float)*n);
    hipMalloc ((void**)&devInfo, sizeof(int));

    hipsolverDnSgeqrf_bufferSize(
        cusolverH,
        m,
        n,
        d_A,
        m,
        &lwork_geqrf);
    hipsolverDnSorgqr_bufferSize(
        cusolverH,
        m,
        n,
        n,
        d_A,
        m,
        d_tau,
        &lwork_orgqr);
    lwork = (lwork_geqrf > lwork_orgqr)? lwork_geqrf : lwork_orgqr;
    hipMalloc((void**)&d_work, sizeof(double)*lwork);
    hipsolverDnSgeqrf(
        cusolverH,
        m,
        n,
        d_A,
        m,
        d_tau,
        d_work,
        lwork,
        devInfo);
    hipsolverDnSorgqr(
        cusolverH,
        m,
        n,
        n,
        d_A,
        m,
        d_tau,
        d_work,
        lwork,
        devInfo);

    hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
     //printf("after geqrf: info_gpu = %d\n", info_gpu);

    if (d_work) hipFree(d_work); d_work = NULL;
    if (devInfo) hipFree(devInfo); devInfo = NULL;
    if (d_tau) hipFree(d_tau); d_tau = NULL;
}

void basicSvd(hipsolverHandle_t cusolverH, hipblasHandle_t cublasH, float *d_B, const int m, const int n, float *d_UT, float *d_S, float *d_V){
    // printf("start svd for m: %d, n:%d\n", m,n);
    float *d_BT = NULL, *d_U = NULL;
    float *d_work = NULL, *d_rwork = NULL;
    int *devInfo = NULL;
    int lwork = 0,  info_gpu = 0;

    cudaStat1 = hipMalloc((void**)&d_BT, sizeof(float)*m*n);
    cudaStat2 = hipMalloc((void**)&d_U, sizeof(float)*m*m);
    cudaStat3 = hipMalloc ((void**)&devInfo, sizeof(int));
    assert(cudaStat1 == hipSuccess);
    assert(cudaStat2 == hipSuccess);
    assert(cudaStat3 == hipSuccess);

//转置B
    cublas_status = hipblasSgeam(cublasH,
                            HIPBLAS_OP_T, HIPBLAS_OP_N, 
                            n, m,
                            &alpha,
                            d_B, m,
                            &beta,
                            d_B, n,
                            d_BT, n);
    assert(cublas_status == HIPSOLVER_STATUS_SUCCESS);
    hipDeviceSynchronize();

    cusolver_status = hipsolverDnDgesvd_bufferSize(
        cusolverH,
        n,
        m,
        &lwork );
    assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS);

    cudaStat1 = hipMalloc((void**)&d_work , sizeof(float)*lwork);
    // cout << "svd sapce: " << sizeof(float)*lwork<<endl;
    assert(hipSuccess == cudaStat1);

    signed char jobu = 'S'; // all m columns of U
    signed char jobvt = 'S'; // all n columns of VT
    // cout << "n: " << n <<" m:" << m <<endl;
    cusolver_status = hipsolverDnSgesvd(
        cusolverH,
        jobu,
        jobvt,
        n,
        m,
        d_BT,
        n,
        d_S,
        d_V,
        n,  // ldu
        d_U,
        m, // ldvt,
        d_work,
        lwork,
        d_rwork,
        devInfo);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    assert(hipSuccess == cudaStat1);

    cudaStat1 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);
    // printf("after gesvd: info_gpu = %d\n", info_gpu);
    assert(0 == info_gpu);

//转置U 给UT
    cublas_status = hipblasSgeam(cublasH,
                            HIPBLAS_OP_T, HIPBLAS_OP_N, 
                            m, m,
                            &alpha,
                            d_U, m,
                            &beta,
                            d_U, m,
                            d_UT, m);
    assert(cublas_status == HIPSOLVER_STATUS_SUCCESS);
    hipDeviceSynchronize();

    if(d_BT) hipFree(d_BT); d_BT = NULL;
    if(d_U) hipFree(d_U); d_U = NULL;
    if(d_work) hipFree(d_work); d_work = NULL;
    if(devInfo) hipFree(devInfo); devInfo = NULL;
    if(d_rwork) hipFree(d_rwork); d_rwork = NULL;
}


void basicRandSVD_LU(hipsolverHandle_t cusolverH, hipblasHandle_t cublasH, float *d_A, 
    const int m, const int n, const int k, const int s,const int p, hipblasGemmAlgo_t algo,
    float *d_U, float *d_SVT){
    int ks = k + s;
    float time = 0;

///获得随机正太分布矩阵B
    // timer1.start();
    // cout << "m: " <<m<<"  n: "<<n <<  "  ks: "<<ks<<endl;
    hiprandGenerator_t gen;
    float *d_B, *d_C, *d_UT, *d_W;
    half *d_Ah, *d_Bh, *d_Ch, *d_UTh;
    cudaStat1 = hipMalloc((void**)&d_B, sizeof(float)*n*ks); //d_V
    cudaStat2 = hipMalloc((void**)&d_C, sizeof(float)*m*ks); //d_U
    cudaStat3 = hipMalloc((void**)&d_UT, sizeof(float)*ks*ks);
    cudaStat4 = hipMalloc((void**)&d_Ah, sizeof(half)*m*n);
    cudaStat5 = hipMalloc((void**)&d_Bh, sizeof(half)*n*ks);
    cudaStat6 = hipMalloc((void**)&d_Ch, sizeof(half)*m*ks);
    cudaStat7 = hipMalloc((void**)&d_UTh, sizeof(half)*ks*ks);
    assert(cudaStat1 == hipSuccess);
    assert(cudaStat2 == hipSuccess);
    assert(cudaStat3 == hipSuccess);
    assert(cudaStat4 == hipSuccess);
    assert(cudaStat5 == hipSuccess);
    assert(cudaStat6 == hipSuccess);
    assert(cudaStat7 == hipSuccess);
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
    hiprandGenerateNormal(gen, d_B, n*ks, 0, 1); 

    assert(cudaStat1 == hipSuccess);
///矩阵A 乘以B 获得矩阵C
    cublas_status = hipblasGemmEx(cublasH,
                           HIPBLAS_OP_N, 
                           HIPBLAS_OP_N,
                           m,
                           ks,
                           n,
                           &alpha,
                           d_A, 
                           HIP_R_32F, 
                           m,
                           d_B, 
                           HIP_R_32F, 
                           n,
                           &beta,
                           d_C, 
                           HIP_R_32F, 
                           m,
                           HIP_R_32F, 
                           algo);
    assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
    
    basicQR(cusolverH, d_C, m, ks);

    for(int i = 0; i < p; ++i){
    // B = A'*Q (d_C)
        
        cublas_status = hipblasGemmEx(cublasH,
                           HIPBLAS_OP_T, 
                           HIPBLAS_OP_N,
                           n,
                           ks,
                           m,
                           &alpha,
                           d_A,
                           HIP_R_32F,
                           m,
                           d_C,
                           HIP_R_32F,
                           m,
                           &beta,
                           d_B,
                           HIP_R_32F,
                           n,
                           HIP_R_32F,
                           algo);
        assert(cublas_status == HIPBLAS_STATUS_SUCCESS);

    // Q(d_B) = QR(B)
        basicQR(cusolverH, d_B, n, ks);

    // C = A * Q (d_B)
        cublas_status = hipblasGemmEx(cublasH,
                           HIPBLAS_OP_N, 
                           HIPBLAS_OP_N,
                           m, 
                           ks, 
                           n,
                           &alpha,
                           d_A, 
                           HIP_R_32F, 
                           m, 
                           d_B, 
                           HIP_R_32F, 
                           n,
                           &beta,
                           d_C, 
                           HIP_R_32F, 
                           m,
                           HIP_R_32F, 
                           algo);
        assert(cublas_status == HIPBLAS_STATUS_SUCCESS);

    // Q(d_C) = QR(C)
        basicQR(cusolverH, d_C, m, ks);
        cudaStat1 = hipDeviceSynchronize();
        assert(cudaStat1 == hipSuccess);
    }
    if(d_Bh) hipFree(d_Bh);d_Bh=NULL;

///d_B = d_C^T(Q^T)*d_A
    cublas_status = hipblasGemmEx(cublasH,
                           HIPBLAS_OP_T, 
                           HIPBLAS_OP_N,
                           ks, 
                           n, 
                           m,
                           &alpha,
                           d_C, 
                           HIP_R_32F, 
                           m,
                           d_A, 
                           HIP_R_32F, 
                           m,
                           &beta,
                           d_B, 
                           HIP_R_32F, 
                           ks,
                           HIP_R_32F, 
                           algo);
    assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
    if(d_Ah) hipFree(d_Ah);d_Ah=NULL;
    // printMatrix_Device(ks,n,d_B,10, "B");
    // printf("ks:%d n:%d for B\n", ks, n);
    hipMalloc ((void**)&d_W, sizeof(dt) * ks);
///对B进行分解
    // GPUTimer timerSVD;
    // timerSVD.start();
    hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, ks, ks, n, &alpha, d_B, ks, d_B, ks, &beta, d_U, ks);
    // printMatrix_Device(ks, ks, d_U, 10, "BBT");
    // basicSvd(cusolverH, cublasH, d_B, ks, n, d_UT, d_S, d_V);
    basicEig(cusolverH, cublasH, d_U, ks, d_W);
    // printMatrix_Device(ks, ks, d_U, 10, "U");
    cudaStat1 = hipDeviceSynchronize();
    assert(hipSuccess == cudaStat1);
    if (d_W) hipFree(d_W);
    dim3 threads(512,1,1);
    dim3 blocksUinv((ks*ks+512-1)/512,1,1);
    // dim3 blocksUinv((m*m+512-1)/512,1,1);  TODO 验证
    matrixInvertColumn<<<blocksUinv, threads>>>(d_U, d_UT, ks, ks);

    cublas_status = hipblasGemmEx(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N,
                           ks, n, ks,
                           &alpha, d_UT, HIP_R_32F, ks,
                           d_B, HIP_R_32F, ks,
                           &beta, d_SVT, HIP_R_32F, ks,
                           HIP_R_32F, algo);
    assert(cublas_status == HIPBLAS_STATUS_SUCCESS);

/// U = Q*UT
    cublas_status = hipblasGemmEx(cublasH,
                           HIPBLAS_OP_N, 
                           HIPBLAS_OP_N,
                           m,
                           ks,
                           ks,
                           &alpha,
                           d_C, 
                           HIP_R_32F, 
                           m,
                           d_UT,
                           HIP_R_32F, 
                           ks,
                           &beta,
                           d_U, 
                           HIP_R_32F, 
                           m,
                           HIP_R_32F, 
                           algo);
    assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
    // printf("\n***********************\n time svd: %f \n*********************\n",  timerSVD.seconds());
    // printMatrix_Device(m,ks,d_U,ks,"U");
    // time = timer1.seconds();
    // printf("\n---------------------------------\nrandom b time:%f\n---------------------------------\n",time);
    // printMatrix_Device(m,ks,d_C,m, "C");
    if(d_B) hipFree(d_B); d_B = NULL;
    if(d_C) hipFree(d_C); d_C = NULL;
    if(d_Ch) hipFree(d_Ch); d_Ch = NULL;
    if(d_UT) hipFree(d_UT); d_UT = NULL;
    if(d_UTh) hipFree(d_UTh); d_UTh = NULL;
}


float ttdec_gpu_float(float* h_tensor, int* h_n_array, int* h_r_array,double *eps, float* singleError, float* singleComRatio ){
    // cout << "algo: " << algo <<endl;
    GPUTimer timer0;
    timer0.start();

    // GPUTimer timer1;
    // timer1.start();

    hipsolverHandle_t cusolverH = NULL;
    hipblasHandle_t cublasH = NULL;

    dt *d_A = NULL;  /* device copy of A */
    dt *d_U = NULL;  /* left singular vectors */

    dt *h_G1 = NULL;

    const int m = h_n_array[0];  //1500
    const int n = h_n_array[1] * h_n_array[2];  //1500 * 1500 = 2250000
    int k = h_r_array[1];
    const int s = 0;
    const int p = 1;
    int ks = k+s;
  
    /* step 1: create cusolver handle, bind a stream */
    cusolver_status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

    cublas_status = hipblasCreate(&cublasH);
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);
        
    cublas_status = hipblasSetMathMode(cublasH, mathMode);
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

    
    cudaStat1 = hipMalloc ((void**)&d_A  , sizeof(float) * m * n);
    cudaStat2 = hipMalloc ((void**)&d_U  , sizeof(float) * m * (ks));
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    // GPUTimer timer;
    // timer.start();
    cudaStat1 = hipMemcpyAsync(d_A, h_tensor, sizeof(dt)*m*n, hipMemcpyHostToDevice);
    cudaStat2 = hipDeviceSynchronize();
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);


///计算rsvd_LU获得矩阵 [U SVT]
    dt *d_A2 = NULL;
    cudaStat1 =hipMalloc((void**)&d_A2, sizeof(dt)*ks*n); 

    basicRandSVD_LU(cusolverH, cublasH, d_A, m, n, k, s, p, algo, d_U, d_A2);

    int truncat_number = h_r_array[1];
    

///根绝截断值将核G1复制到内存 
    cudaStat1 = hipHostAlloc((void**)&h_G1, sizeof(dt)*m*truncat_number, 0);
    cudaStat2 = hipMemcpyAsync(h_G1, d_U, sizeof(dt)*m*truncat_number, hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);

    dt *d_G3 = NULL;
    dt *h_G2 = NULL;
    dt *h_G3 = NULL;
    dt *d_U2 = NULL;
    const int m2 = h_n_array[1] * h_r_array[1];  // 56*1000
    const int n2 = h_n_array[2];  // 1000
    k = h_r_array[2];
    truncat_number = h_r_array[2];
    // cout << "truncat_number:" << truncat_number << endl;
    ks = k + s;

 /* step 1: create cusolver handle, bind a stream */
    cudaStat1 = hipMalloc ((void**)&d_U2   , sizeof(dt)*m2*ks); 
    cudaStat2 = hipMalloc ((void**)&d_G3   , sizeof(dt)*n2*truncat_number); 
    cudaStat3 = hipHostAlloc((void**)&h_G2 , sizeof(dt)*m2*truncat_number, 0); //56*1000*56
    cudaStat4 = hipHostAlloc((void**)&h_G3 , sizeof(dt)*n2*truncat_number, 0); //1000*56
    assert(cudaStat1 == hipSuccess);
    assert(cudaStat2 == hipSuccess);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat4);

    basicRandSVD_LU(cusolverH, cublasH, d_A2, m2, n2, k, s, p, algo, d_U2, d_G3);

    if(d_A2) hipFree(d_A2); d_A2 = NULL;
    
    cudaStat1 = hipMemcpyAsync(h_G2, d_U2, sizeof(dt)*m2*truncat_number, hipMemcpyDeviceToHost);
    cudaStat2 = hipMemcpyAsync(h_G3, d_G3, sizeof(dt)*n2*truncat_number, hipMemcpyDeviceToHost);
    cudaStat3 = hipDeviceSynchronize();
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);


if(CalculateError){
    long long calAllNumber = (long long) h_n_array[0]*(long long)h_n_array[1]*(long long)h_n_array[2];
    printf("calAllNumber : %lld\n", calAllNumber);
    double re = 0.0, before = 0.0;
    const float h_one = 1;
    const float h_minus_one = -1;
    before = norm2HH(h_tensor, calAllNumber);

    dim3 threads(1024,1,1);
    dim3 block3n((calAllNumber+1024-1)/1024,1,1); 
    float *d_coreG2G3 = NULL, *h_A = NULL;//, *d_coreG1G2G3 = NULL;
    cudaStat1 = hipMalloc((void**)&d_coreG2G3, sizeof(float)*m2*n2); // 56*1000*1000
    cudaStat2 = hipHostAlloc((void**)&h_A,sizeof(dt)*calAllNumber,0);
    // hipMalloc((void**)&d_coreG1G2G3, sizeof(float)*calAllNumber); //1000*1000*1000
    cudaStat3 = hipMemcpy(d_A, h_tensor, sizeof(dt)*calAllNumber, hipMemcpyHostToDevice);
    assert(cudaStat1 == hipSuccess);
    assert(cudaStat2 == hipSuccess);
    assert(cudaStat3 == hipSuccess);


    // printMatrix_Device(10, 10, d_U2, 10, "G2");
    // printMatrix_Device(10, 10, d_G3, 10, "G3");
    cublas_status = hipblasGemmEx(cublasH,
                           HIPBLAS_OP_N, 
                           HIPBLAS_OP_N,
                           m2, n2, truncat_number,
                           &alpha,
                           d_U2, 
                           HIP_R_32F, 
                           m2,
                           d_G3, 
                           HIP_R_32F, 
                           truncat_number,
                           &beta,
                           d_coreG2G3, 
                           HIP_R_32F, 
                           m2,
                           HIP_R_32F, 
                           algo);
    assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
    cublas_status = hipblasGemmEx(cublasH,
                           HIPBLAS_OP_N, 
                           HIPBLAS_OP_N,
                           h_n_array[0],  h_n_array[1]*h_n_array[2], h_r_array[1],
                           &h_minus_one,
                           d_U, 
                           HIP_R_32F, 
                           h_n_array[0],
                           d_coreG2G3, 
                           HIP_R_32F, 
                           h_r_array[1],
                           &h_one,
                           d_A, 
                           HIP_R_32F, 
                           h_n_array[0],
                           HIP_R_32F, 
                           algo);
    assert(cublas_status == HIPBLAS_STATUS_SUCCESS);

    hipDeviceSynchronize();

    cudaStat2 = hipMemcpy(h_A, d_A, sizeof(float) * calAllNumber, hipMemcpyDeviceToHost); 
    assert(hipSuccess == cudaStat2);

    re = norm2HH(h_A, calAllNumber);
    cout << "re: "<< re << " before: " << before <<endl;
    printf("\n******************************\n error rate: %E \n*****************************\n", re/before);

    if(d_coreG2G3) hipFree(d_coreG2G3); d_coreG2G3=NULL;
    if (h_A) hipHostFree(h_A); h_A = NULL;
}


    if(d_A) hipFree(d_A); d_A = NULL;
    if(d_G3) hipFree(d_G3); d_G3 = NULL;
    if(d_U2) hipFree(d_U2); d_U2 = NULL;

    if(d_U) hipFree(d_U); d_U = NULL;

    if (h_G1) hipHostFree(h_G1); h_G1 = NULL;
    if (h_G2) hipHostFree(h_G2); h_G2 = NULL;
    if (h_G3) hipHostFree(h_G3); h_G3 = NULL;

    if (cublasH ) hipblasDestroy(cublasH); cublasH = NULL;
    if (cusolverH) hipsolverDnDestroy(cusolverH); cusolverH = NULL;
   
    float time = timer0.seconds();
    // cout << "----------------------------------\nall run time: " << time << endl;
    return time;
}