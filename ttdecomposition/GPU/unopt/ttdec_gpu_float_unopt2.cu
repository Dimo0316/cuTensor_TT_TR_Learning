#include "hip/hip_runtime.h"
#include "head.h"

__global__ void tranforArrayToDiagonalMatrix(float* array, float* diagMatrix, int n){
    long long i = blockIdx.x * blockDim.x + threadIdx.x;
    const long long temp = blockDim.x*gridDim.x;
    while(i < n*n){
        long row = i/n;
        long col = i%n;
        if(row == col)
            diagMatrix[i] = array[row];
        else
            diagMatrix[i] = 0;
        i+=temp;
    }
    __syncthreads();
}

__global__ void hardm(dt *M,dt *N,dt *res,long  m){
     long i = blockIdx.x*blockDim.x+threadIdx.x;
    if(i<m){
        res[m-i] = M[i]*N[i];
    }
    __syncthreads();
}

float ttdec_gpu_float_unopt(float* h_tensor, int* h_n_array, int* h_r_array, double eps){
// /计算第一部分svd
    dt time0 = 0.0f;
    GPUTimer timer0;
    timer0.start();
    hipsolverHandle_t cusolverH = NULL;
    hipblasHandle_t cublasH = NULL;
    // warmup<<<1,1>>>();
    // hipDeviceSynchronize();
    hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
    hipsolverStatus_t cusolver_status  = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    hipError_t cudaStat4 = hipSuccess;
    hipError_t cudaStat5 = hipSuccess;
    hipError_t cudaStat6 = hipSuccess;


    cusolver_status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

    cublas_status = hipblasCreate(&cublasH);
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

    // hipblasSetMathMode(cublasH,HIPBLAS_TENSOR_OP_MATH);
    hipblasSetMathMode(cublasH,HIPBLAS_DEFAULT_MATH);

    dt *d_A = NULL;  /* device copy of A */
    dt *d_AT= NULL;
    dt *d_S = NULL;  /* singular values */
    dt *d_U = NULL;  /* left singular vectors */
    dt *d_VT = NULL;  /* right singular vectors */
    dt *d_G1 = NULL;
    dt *h_G1 = NULL;
    int *devInfo = NULL;
    dt *d_work = NULL;
    dt *d_rwork = NULL;
    // dt *d_W = NULL;  // W = S*VT

    int m = h_n_array[0];
    int n = h_n_array[1] * h_n_array[2];
    int lwork = 0;
    int info_gpu = 0;
    float norm_d_S = 0.0f;
    float truncated_Value = 0.0f;
    // const dt h_one = 1;
    const dt alpha = 1.0;
    const float ep = eps/1.414214;
    const dt beta = 0.0;

    cudaStat1 = hipMalloc ((void**)&d_A   , sizeof(dt)*m*n);
    cudaStat2 = hipMalloc ((void**)&d_AT  , sizeof(dt)*m*n);
    cudaStat3 = hipMemcpy (d_AT, h_tensor, sizeof(dt)*m*n, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);

    // cout << " m1:" << m << "  n1:" << n << endl;
    
    //printMatrix_Device(m, n, d_AT, m, "A的转置前");
///判断是否要转置
    if(m < n){
        int temp = m;
        m = n;
        n = temp;  // (m > n)
        cublas_status = hipblasSgeam(cublasH,
                                HIPBLAS_OP_T, HIPBLAS_OP_N, 
                                m, n,
                                &alpha,
                                d_AT, n,
                                &beta,
                                d_A, m,
                                d_A, m);
    }
    else{
        cublas_status = hipblasSgeam(cublasH,
                                HIPBLAS_OP_N, HIPBLAS_OP_N, 
                                m, n,
                                &alpha,
                                d_AT, m,
                                &beta,
                                d_AT, n,
                                d_A, m);
    }
    assert(cublas_status == HIPSOLVER_STATUS_SUCCESS);
    if (d_AT) hipFree(d_AT);
    hipDeviceSynchronize();

    int lda = m;  

/* step 3: copy A and B to device */
    
    cudaStat2 = hipMalloc ((void**)&d_S   , sizeof(dt)*n);
    cudaStat3 = hipMalloc ((void**)&d_U   , sizeof(dt)*m*n); //有问题
    cudaStat4 = hipMalloc ((void**)&d_VT   , sizeof(dt)*n*n);
    cudaStat5 = hipMalloc ((void**)&devInfo, sizeof(int));
    // cudaStat1 = hipMalloc ((void**)&d_W  , sizeof(dt)*lda*n);
    cudaStat1 = hipMalloc ((void**)&d_G1  , sizeof(dt)*n*n);
    // if(cudaStat3 == hipErrorOutOfMemory)
    //     cout << " hipErrorOutOfMemory "<<endl;
    // if(cudaStat3 == hipErrorInvalidValue)
    //     cout << " hipErrorInvalidValue "<<endl;
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);//有问题
    assert(hipSuccess == cudaStat4);
    assert(hipSuccess == cudaStat5);
    assert(hipSuccess == cudaStat1);

    // cudaStat1 = hipHostAlloc((void**)&h_S   , sizeof(dt)*n  , 0);
    
    // cudaStat3 = hipHostAlloc((void**)&h_U   , sizeof(dt)*m*n, 0);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);

    // cout<< " m:" << m << " n:" << n  << " lda:" << lda <<endl;

// step 3: query working space of SVD
    cusolver_status = hipsolverDnSgesvd_bufferSize(
        cusolverH,
        m,
        n,
        &lwork );
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    // cout << "part one work space: " <<sizeof(dt)*lwork << endl;
    cudaStat1 = hipMalloc((void**)&d_work , sizeof(dt)*lwork);
    assert(hipSuccess == cudaStat1);
    // printMatrix_Device(m, n, d_A, m, "A");
    dt time = 0.0f;
    GPUTimer timer;
    timer.start();
    /* step 5: compute SVD */
    signed char jobu = 'S'; // all m columns of U
    signed char jobvt = 'S'; // all n columns of VT
    // cout << "m:" << m <<" n:" <<n<< " lda:" <<lda<<endl;
    cusolver_status = hipsolverDnSgesvd(
        cusolverH,
        jobu,
        jobvt,
        m,
        n,
        d_A,
        m,
        d_S,
        d_U,
        m,  // ldu
        d_VT,
        n, // ldvt,
        d_work,
        lwork,
        d_rwork,
        devInfo);
    cudaStat1 = hipDeviceSynchronize();
    // if(cusolver_status == HIPSOLVER_STATUS_NOT_INITIALIZED)
    //     cout << "HIPSOLVER_STATUS_NOT_INITIALIZED" << endl;
    // if(cusolver_status == HIPSOLVER_STATUS_INVALID_VALUE)
    //     cout << "HIPSOLVER_STATUS_INVALID_VALUE" << endl;
    // if(cusolver_status == HIPSOLVER_STATUS_ARCH_MISMATCH)
    //     cout << "HIPSOLVER_STATUS_ARCH_MISMATCH" << endl;  
    // if(cusolver_status == HIPSOLVER_STATUS_INTERNAL_ERROR)
    //     cout << "HIPSOLVER_STATUS_INTERNAL_ERROR" << endl;

    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    assert(hipSuccess == cudaStat1);

    // cout << "m:" << m << " n:" << n <<" lda:" << lda <<endl;
    
    // printMatrix_Device(n, 1, d_S, n, "S");
    // printMatrix_Device(m, n, d_U, m, "U");
    // printMatrix_Device(n, n, d_VT, n, "VT");

    time = timer.seconds();
    cout << "----------------------------------\npart one svd run time: " << time << endl;
    // printMatrix_Device(n, n, d_G1, n, "G1");
    // printMatrix_Device(n,1,d_S, n,"d_S");
    cudaStat4 = hipMemcpyAsync(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    assert(hipSuccess == cudaStat4);
    printf("after gesvd: info_gpu = %d\n", info_gpu);
    assert(0 == info_gpu);
   
    if (d_A) hipFree(d_A);
    // if (d_S) hipFree(d_S);
    // if (d_U) hipFree(d_U);
    if (devInfo) hipFree(devInfo);
    if (d_work ) hipFree(d_work);
    if (d_rwork) hipFree(d_rwork);

    dim3 threads1024(1024,1,1);
    dim3 blocks1024((n+1024-1)/1024,1,1);

//计算二范数。获得截断值
    dt *d_Shardm = NULL;
    cudaStat1 = hipMalloc((void**)&d_Shardm, sizeof(dt)*n);
    assert(cudaStat1 == hipSuccess);
    cublas_status = hipblasSnrm2(cublasH, n,
                d_S, 1, &norm_d_S);
    assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
    cudaStat1 = hipDeviceSynchronize();
    assert(cudaStat1 == hipSuccess);
    truncated_Value = norm_d_S * ep * norm_d_S * ep;
///获得截断索引 truncat_number 也就是h_r_array
///TODO
    float sumError = 0.0f;
    int truncat_number_test = 0;
    hardm<<<blocks1024,threads1024>>>(d_S,d_S,d_Shardm,n);
    // printMatrix_Device(econ_s,1,d_S,econ_s,"S");
    // printMatrix_Device(econ_s,1,d_Shardm,econ_s,"Shardm");
    cout << "truncated_Value" << truncated_Value << endl;
    while(sumError < truncated_Value){
        cublas_status = hipblasSasum(cublasH, truncat_number_test,
                            d_Shardm, 1, &sumError);
        assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
        // cout << "sumError" << sumError << endl;
        truncat_number_test++;
    }
    // time = timer.seconds();
    // cout << "truncat_number_test:" << econ_s - truncat_number_test + 3 <<endl;
    if(d_Shardm) hipFree(d_Shardm);
    h_r_array[1] = n - truncat_number_test + 2;
    int truncat_number = h_r_array[1];
    cout << "truncat_number:" << truncat_number << endl;

///复制G1到内存 h_G1
    cublas_status = hipblasSgeam(cublasH,
                                HIPBLAS_OP_T, HIPBLAS_OP_N, 
                                n, n,
                                &alpha,
                                d_VT, n,
                                &beta,
                                d_VT, n,
                                d_G1, n);
    assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
    cudaStat6 = hipDeviceSynchronize();
    assert(cudaStat6 == hipSuccess);
    if (d_VT   ) hipFree(d_VT);
    float *d_Struncat = NULL;
///计算中间变量 第二部分
    dt *d_Smatrix = NULL;
    dt *d_AT2 = NULL;
    cudaStat4 = hipHostAlloc((void**)&h_G1  , sizeof(dt)*truncat_number*n, 0);
    cudaStat1 = hipMalloc((void**)&d_Struncat,  sizeof(dt)*truncat_number);
    cudaStat2 = hipMalloc((void**) &d_Smatrix,    sizeof(dt)*truncat_number*truncat_number);
    cudaStat3 = hipMalloc((void**) &d_AT2,         sizeof(dt)*truncat_number*m);
    assert(cudaStat1 == hipSuccess);
    assert(cudaStat2 == hipSuccess);
    assert(cudaStat3 == hipSuccess);
    assert(cudaStat4 == hipSuccess);

    cudaStat5 = hipMemcpyAsync(h_G1, d_G1, sizeof(dt)*truncat_number*n, hipMemcpyDeviceToHost);
    cudaStat6 = hipDeviceSynchronize();
    assert(cudaStat5 == hipSuccess);
    assert(cudaStat6 == hipSuccess);

    cublas_status = hipblasScopy(cublasH, truncat_number,
                            d_S, 1,
                            d_Struncat, 1);
    assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
    cudaStat1 = hipDeviceSynchronize();
    assert(cudaStat1 == hipSuccess);
    if(d_S) hipFree(d_S);
    if (d_G1) hipFree(d_G1);
    
    // printMatrix_Device(n,n,d_Smatrix,n,"Smatrix1");
    GPUTimer timer2;
    timer2.start();
    dim3 thread(512,1,1);
    dim3 blocks((n*n+1024-1)/1024,1,1);

    tranforArrayToDiagonalMatrix<<<blocks,thread>>>(d_Struncat,d_Smatrix,truncat_number);
    cudaStat1 = hipDeviceSynchronize();
    assert(cudaStat1 == hipSuccess);
    // printMatrix_Device(n,n,d_Smatrix,n,"Smatrix");
//TODo
    cublas_status = hipblasGemmEx(cublasH, 
        HIPBLAS_OP_N, HIPBLAS_OP_T,
        truncat_number, m, truncat_number,
        &alpha,
        d_Smatrix, HIP_R_32F, truncat_number,
        d_U, HIP_R_32F, m,
        &beta,
        d_AT2, HIP_R_32F, truncat_number,
        HIP_R_32F, HIPBLAS_GEMM_DEFAULT);
    assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
    cudaStat1 = hipDeviceSynchronize();
    assert(cudaStat1 == hipSuccess);
    time = timer2.seconds();
    cout << "----------------------------------\n part two gemm run time: " << time << endl;
    // printMatrix_Device(n,m,d_AT2,n,"AT2");
    // if(h_U) hipHostFree(h_U);
    // if(h_S) hipHostFree(h_S);
    // if(h_Umatrix) hipHostFree(h_Umatrix);
    // if(h_Smatrix) hipHostFree(h_Smatrix);
    if(d_Smatrix) hipFree(d_Smatrix);
    if(d_S) hipFree(d_S);
    if(d_U) hipFree(d_U);

///计算第三部分svd
    cusolverH = NULL;
    cusolver_status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

    // dt *d_A2= NULL;
    dt *d_S2 = NULL;  /* singular values */
    dt *d_U2 = NULL;  /* left singular vectors */
    dt *d_VT2 = NULL;  /* right singular vectors */
    dt *d_G3 = NULL;
    //dt *h_G2T = NULL;
    dt *h_G2 = NULL;
    dt *h_G3 = NULL;
    int *devInfo2 = NULL;
    dt *d_work2 = NULL;
    dt *d_rwork2 = NULL;
    
    // cudaStat1 = hipMalloc ((void**)&d_A2, sizeof(dt)*m*n);
    assert(cudaStat1 == hipSuccess);
    m = h_n_array[1]*h_r_array[1];
    n = h_n_array[2];

   
    // if (d_AT2) hipFree(d_AT2);
    cudaStat1 =  hipDeviceSynchronize();
    assert(hipSuccess == cudaStat1);
    // printMatrix_Device(m,n,d_A2,m,"A2");
    lda = m;  

/* step 3: copy A and B to device */
    cudaStat1 = hipMalloc ((void**)&d_G3   , sizeof(dt)*n*n);
    cudaStat2 = hipMalloc ((void**)&d_S2   , sizeof(dt)*n);
    cudaStat3 = hipMalloc ((void**)&d_U2   , sizeof(dt)*m*n);
    cudaStat4 = hipMalloc ((void**)&d_VT2   , sizeof(dt)*n*n);
    cudaStat5 = hipMalloc ((void**)&devInfo2, sizeof(int));
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat4);
    assert(hipSuccess == cudaStat5);

    int lwork2 = 0;
// step 3: query working space of SVD
    cusolver_status = hipsolverDnSgesvd_bufferSize(
        cusolverH,
        m,
        n,
        &lwork2 );
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    // cout << "part three work space: " <<sizeof(dt)*lwork2 << endl;
    cudaStat1 = hipMalloc((void**)&d_work2 , sizeof(dt)*lwork2);
    assert(hipSuccess == cudaStat1);
     // cout << "m:" << m << " n:" << n <<" lda:" << lda <<endl;
    time = 0.0f;

    GPUTimer timer3;
    timer3.start();
    cusolver_status = hipsolverDnSgesvd(
        cusolverH,
        jobu,
        jobvt,
        m,
        n,
        d_AT2,
        lda,
        d_S2,
        d_U2,
        m,  // ldu
        d_VT2,
        n, // ldvt,
        d_work2,
        lwork2,
        d_rwork2,
        devInfo2);
    cudaStat1 = hipDeviceSynchronize();
    // if(cusolver_status == HIPSOLVER_STATUS_NOT_INITIALIZED)
    //     cout << "HIPSOLVER_STATUS_NOT_INITIALIZED" << endl;
    // if(cusolver_status == HIPSOLVER_STATUS_INVALID_VALUE)
    //     cout << "HIPSOLVER_STATUS_INVALID_VALUE" << endl;
    // if(cusolver_status == HIPSOLVER_STATUS_ARCH_MISMATCH)
    //     cout << "HIPSOLVER_STATUS_ARCH_MISMATCH" << endl;  
    // if(cusolver_status == HIPSOLVER_STATUS_INTERNAL_ERROR)
    //     cout << "HIPSOLVER_STATUS_INTERNAL_ERROR" << endl;
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    assert(hipSuccess == cudaStat1);
    if(d_AT2) hipHostFree(d_AT2);
    
    // printMatrix_Device(n, 1, d_S2, n, "S2");
    // printMatrix_Device(m, n, d_U2, m, "U2");
    // printMatrix_Device(n, n, d_VT2, n, "VT2");
    time = timer3.seconds();
    cout << "----------------------------------\npart three svd run time: " << time << endl;
    
    //计算获得G3 TODO改成非优化版本
    // cublas_status = hipblasSdgmm(cublasH,
    //                             HIPBLAS_SIDE_LEFT,
    //                             n, n,
    //                             d_V2, n,
    //                             d_S2, 1,
    //                             d_G3, n);
    // assert(statusCublas == HIPBLAS_STATUS_SUCCESS);
    // cudaStat1 = hipDeviceSynchronize();
    // assert(hipSuccess == cudaStat1);

    //计算二范数。获得截断值
    dt *d_Shardm2 = NULL;
    cudaStat1 = hipMalloc((void**)&d_Shardm2, sizeof(dt)*n);
    assert(cudaStat1 == hipSuccess);
    cublas_status = hipblasSnrm2(cublasH, n,
                d_S2, 1, &norm_d_S);
    assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
    cudaStat1 = hipDeviceSynchronize();
    assert(cudaStat1 == hipSuccess);
    truncated_Value = norm_d_S * ep * norm_d_S * ep;
     // cout <<"truncated :" << truncated  << " norm_d_S:" << norm_d_S << endl;
///获得截断索引 truncat_number 也就是h_r_array
///TODO
    sumError = 0.0f;
    truncat_number_test = 0;
    dim3 blocks10242((n+1024-1)/1024,1,1);
    hardm<<<blocks10242,threads1024>>>(d_S2,d_S2,d_Shardm2,n);
    // printMatrix_Device(econ_s2,1,d_S2,econ_s2,"S2");
    // printMatrix_Device(econ_s,1,d_Shardm,econ_s,"Shardm");
    cout << "truncated_Value2" << truncated_Value << endl;
    while(sumError < truncated_Value){
        cublas_status = hipblasSasum(cublasH, truncat_number_test,
                            d_Shardm2, 1, &sumError);
        assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
        // cout << "sumError2" << sumError << endl;
        truncat_number_test++;
    }
    // time = timer2.seconds();
    // cout << "truncat_number_test2:" << econ_s2 - truncat_number_test + 3 <<endl;
    if(d_Shardm2) hipFree(d_Shardm2);

     // cout <<"truncated :" << truncated  << " norm_d_S:" << norm_d_S << endl;
///获得截断索引 truncat_number 也就是h_r_array
///TODO
    h_r_array[2] = n - truncat_number_test + 2;
    truncat_number = h_r_array[2];
    cout << "truncat_number:" << truncat_number << endl;
    
    dt *d_Smatrix2 = NULL, *d_Struncat2 = NULL;//, *d_Vtruncat2 = NULL;
    cudaStat1 = hipMalloc((void**)&d_Struncat2, sizeof(dt)*truncat_number);
    cudaStat2 = hipMalloc((void**) &d_Smatrix2, sizeof(dt)*truncat_number*truncat_number);
    cudaStat3 = hipHostAlloc((void**)&h_G2  , sizeof(dt)*m*truncat_number, 0);
    cudaStat4 = hipHostAlloc((void**)&h_G3  , sizeof(dt)*n*truncat_number, 0);
    cudaStat5 = hipDeviceSynchronize();
    assert(cudaStat1 == hipSuccess);
    //assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    assert(cudaStat4 == hipSuccess);
    assert(cudaStat5 == hipSuccess);


    ///截断S2
    cublas_status = hipblasScopy(cublasH, truncat_number,
                            d_S2, 1,
                            d_Struncat2, 1);
    assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
    cudaStat1 = hipDeviceSynchronize();
    assert(cudaStat1 == hipSuccess);
    if(d_S2) hipFree(d_S2);

    GPUTimer timer4; 
    timer4.start();
    dim3 thread2(512,1,1);
    dim3 blocks2((n*n+1024-1)/1024,1,1);
    tranforArrayToDiagonalMatrix<<<blocks2,thread2>>>(d_Struncat2,d_Smatrix2,truncat_number);
    cudaStat1 = hipDeviceSynchronize();
    assert(cudaStat1 == hipSuccess);

    cublas_status = hipblasGemmEx(cublasH, 
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        truncat_number, n, truncat_number,
        &alpha,
        d_Smatrix2, HIP_R_32F, truncat_number,
        d_VT2, HIP_R_32F, truncat_number,
        &beta,
        d_G3, HIP_R_32F, truncat_number,
        HIP_R_32F, HIPBLAS_GEMM_DEFAULT);
    assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
    cudaStat1 = hipDeviceSynchronize();
    assert(cudaStat1 == hipSuccess);
    time = timer4.seconds();
    cout << "----------------------------------\n part four gemm run time: " << time << endl;

    // printMatrix_Device(n,n,d_G3,n,"G3");

    cudaStat4 = hipMemcpyAsync(&info_gpu, devInfo2, sizeof(int), hipMemcpyDeviceToHost);
    cudaStat3 = hipMemcpyAsync(h_G2, d_U2, sizeof(dt)*m*truncat_number, hipMemcpyDeviceToHost);
    cudaStat2 = hipMemcpyAsync(h_G3, d_G3, sizeof(dt)*n*truncat_number, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    assert(hipSuccess == cudaStat4);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat2);


    printf("after gesvd: info_gpu = %d\n", info_gpu);
    assert(0 == info_gpu);
    

    // if(d_A) hipFree(d_A);
    // if(d_G3) hipFree(d_G3);
    // if(d_G1) hipFree(d_G1);


    if (d_S2) hipFree(d_S2);
    if (d_U2) hipFree(d_U2);
    if (d_VT2) hipFree(d_VT2);
    if (d_G3) hipFree(d_G3);
    if (devInfo2) hipFree(devInfo2);
    if (d_work2 ) hipFree(d_work2);
    if (d_rwork2) hipFree(d_rwork2);
    if (d_Smatrix2) hipFree(d_Smatrix2);
    //printf("step2 - calculate mid success--------2*12--------- \n");
    
    //if (h_G2T) hipHostFree(h_G2T);
    if (h_G1) hipHostFree(h_G1);
    if (h_G2) hipHostFree(h_G2);
    if (h_G3) hipHostFree(h_G3);
    if (cublasH ) hipblasDestroy(cublasH);
    if (cusolverH) hipsolverDnDestroy(cusolverH);
    
    time0 = timer0.seconds();
    cout << "----------------------------------\nfinish all: " << endl;
    return time0;

}