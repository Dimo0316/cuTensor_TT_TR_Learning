#include "head.h"

void ttdec_gpu_float_unopt(float* h_tensor, int* h_n_array, double eps){
///计算第一部分svd
    hipsolverHandle_t cusolverH = NULL;
    hipblasHandle_t cublasH = NULL;

    hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
    hipsolverStatus_t cusolver_status  = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    hipError_t cudaStat4 = hipSuccess;
    hipError_t cudaStat5 = hipSuccess;


    cusolver_status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

    cublas_status = hipblasCreate(&cublasH);
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

    dt *d_A = NULL;  /* device copy of A */
    dt *d_AT= NULL;
    dt *d_S = NULL;  /* singular values */
    dt *h_S = NULL;
    dt *d_U = NULL;  /* left singular vectors */
    dt *d_VT = NULL;  /* right singular vectors */
    dt *d_G1 = NULL;
    dt *h_G1 = NULL;
    dt *h_U = NULL;
    int *devInfo = NULL;
    dt *d_work = NULL;
    dt *d_rwork = NULL;
    dt *d_W = NULL;  // W = S*VT

    int m = h_n_array[0];
    int n = h_n_array[1] * h_n_array[2];
    int lwork = 0;
    int info_gpu = 0;
    // const dt h_one = 1;
    // const dt h_minus_one = -1;
    const dt alpha = 1.0;
    const dt beta = 0.0;

    cudaStat1 = hipMalloc ((void**)&d_A   , sizeof(dt)*m*n);
    cudaStat2 = hipMalloc ((void**)&d_AT  , sizeof(dt)*m*n);
    cudaStat3 = hipMemcpy (d_AT, h_tensor, sizeof(dt)*m*n, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    //cout << " m1:" << m << "  n1:" << n << endl;
    
    //printMatrix_Device(m, n, d_AT, m, "A的转置前");
///判断是否要转置
    if(m < n){
        int temp = m;
        m = n;
        n = temp;  // (m > n)
        cublas_status = hipblasSgeam(cublasH,
                                HIPBLAS_OP_T, HIPBLAS_OP_N, 
                                m, n,
                                &alpha,
                                d_AT, n,
                                &beta,
                                d_A, m,
                                d_A, m);
    }
    else{
        cublas_status = hipblasSgeam(cublasH,
                                HIPBLAS_OP_N, HIPBLAS_OP_N, 
                                m, n,
                                &alpha,
                                d_AT, m,
                                &beta,
                                d_AT, n,
                                d_A, m);
    }
    assert(cublas_status == HIPSOLVER_STATUS_SUCCESS);
    if (d_AT) hipFree(d_AT);
    hipDeviceSynchronize();
    // printMatrix_Device(m, n, d_A, m, "A");

    int lda = m;  // 1500

/* step 3: copy A and B to device */
    
    cudaStat2 = hipMalloc ((void**)&d_S   , sizeof(dt)*n);
    cudaStat3 = hipMalloc ((void**)&d_U   , sizeof(dt)*m*m); //有问题
    cudaStat4 = hipMalloc ((void**)&d_VT   , sizeof(dt)*n*n);
    cudaStat5 = hipMalloc ((void**)&devInfo, sizeof(int));
    cudaStat1 = hipMalloc ((void**)&d_W  , sizeof(dt)*lda*n);
    cudaStat1 = hipMalloc ((void**)&d_G1  , sizeof(dt)*n*n);
    if(cudaStat3 == hipErrorOutOfMemory)
        cout << " hipErrorOutOfMemory "<<endl;
    if(cudaStat3 == hipErrorInvalidValue)
        cout << " hipErrorInvalidValue "<<endl;
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);//有问题
    assert(hipSuccess == cudaStat4);
    assert(hipSuccess == cudaStat5);
    assert(hipSuccess == cudaStat1);

    cudaStat1 = hipHostAlloc((void**)&h_S   , sizeof(dt)*n  , 0);
    cudaStat2 = hipHostAlloc((void**)&h_G1  , sizeof(dt)*n*n, 0);
    cudaStat3 = hipHostAlloc((void**)&h_U   , sizeof(dt)*m*m, 0);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);

    // cout<< " m:" << m << " n:" << n  << " lda:" << lda <<endl;


// step 3: query working space of SVD
    cusolver_status = hipsolverDnSgesvd_bufferSize(
        cusolverH,
        m,
        n,
        &lwork );
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    cout << "part one work space: " <<sizeof(dt)*lwork << endl;
    cudaStat1 = hipMalloc((void**)&d_work , sizeof(dt)*lwork);
    assert(hipSuccess == cudaStat1);

    dt time = 0.0f;
    GPUTimer timer;
    timer.start();
    /* step 5: compute SVD */
    signed char jobu = 'A'; // all m columns of U
    signed char jobvt = 'A'; // all n columns of VT
    cusolver_status = hipsolverDnSgesvd(
        cusolverH,
        jobu,
        jobvt,
        m,
        n,
        d_A,
        lda,
        d_S,
        d_U,
        lda,  // ldu
        d_VT,
        n, // ldvt,
        d_work,
        lwork,
        d_rwork,
        devInfo);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    assert(hipSuccess == cudaStat1);

    time = timer.seconds();
    cout << "----------------------------------\npart one svd run time: " << time << endl;
///复制G1到内存 h_G1
    cublas_status = hipblasSgeam(cublasH,
                                HIPBLAS_OP_T, HIPBLAS_OP_N, 
                                n, n,
                                &alpha,
                                d_VT, n,
                                &beta,
                                d_VT, n,
                                d_G1, n);
    assert(cublas_status == HIPBLAS_STATUS_SUCCESS);

    // printMatrix_Device(n,1,d_S, n,"d_S");

    cudaStat4 = hipMemcpyAsync(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    cudaStat5 = hipMemcpyAsync(h_S, d_S, sizeof(dt)*n, hipMemcpyDeviceToHost);
    cudaStat3 = hipMemcpyAsync(h_G1, d_G1, sizeof(dt)*n*n, hipMemcpyDeviceToHost);
    cudaStat2 = hipMemcpyAsync(h_U, d_U, sizeof(dt)*m*m, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    assert(hipSuccess == cudaStat4);
    assert(hipSuccess == cudaStat5);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat2);

    printf("after gesvd: info_gpu = %d\n", info_gpu);
    assert(0 == info_gpu);
    // printf("=====\n");

//     // step 6: |A - U*S*VT|
//     // W = S*VT
//     cublas_status = hipblasSdgmm(
//         cublasH,
//         HIPBLAS_SIDE_LEFT,
//         n,
//         n,
//         d_VT,
//         lda,
//         d_S,
//          1,
//         d_W,
//         lda);
//     assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

// // A := -U*W + A
//     cudaStat1 = hipMemcpy(d_A, h_tensor, sizeof(dt)*lda*n, hipMemcpyHostToDevice);
//     assert(hipSuccess == cudaStat1);
//     cublas_status = hipblasSgemm(
//         cublasH,
//         HIPBLAS_OP_N, // U
//         HIPBLAS_OP_N, // W
//         m, // number of rows of A
//         n, // number of columns of A
//         n, // number of columns of U 
//         &h_minus_one, /* host pointer */
//         d_U, // U
//         lda,
//         d_W, // W
//         lda,
//         &h_one, /* hostpointer */
//         d_A,
//         lda);
//     assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

//     dt dR_fro = 0.0;
//     cublas_status = hipblasSnrm2(
//         cublasH, lda*n, d_A, 1, &dR_fro);
//     assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

//     printf("|A - U*S*VT| = %E \n", dR_fro);
    if (d_G1) hipFree(d_G1);
    if (d_A) hipFree(d_A);
    if (d_S) hipFree(d_S);
    if (d_U) hipFree(d_U);
    if (d_VT   ) hipFree(d_VT);
    if (devInfo) hipFree(devInfo);
    if (d_work ) hipFree(d_work);
    if (d_rwork) hipFree(d_rwork);
    //if (d_W    ) hipFree(d_W);

    
///计算中间变量 第二部分
    dt *h_Smatrix = NULL;
    dt *d_Smatrix = NULL;
    dt *h_Umatrix = NULL;
    dt *d_Umatrix = NULL;
    dt *d_AT2 = NULL;
    
    cudaStat1 = hipHostAlloc((void**)&h_Smatrix, sizeof(dt)*n*n, 0);
    cudaStat2 = hipHostAlloc((void**)&h_Umatrix, sizeof(dt)*m*n, 0);
    cudaStat3 = hipMalloc((void**) &d_Smatrix,    sizeof(dt)*n*n);
    cudaStat4 = hipMalloc((void**) &d_Umatrix,    sizeof(dt)*m*n);
    cudaStat5 = hipMalloc((void**) &d_AT2,         sizeof(dt)*n*m);
    hipDeviceSynchronize();
    assert(cudaStat1 == hipSuccess);
    assert(cudaStat2 == hipSuccess);
    assert(cudaStat3 == hipSuccess);
    assert(cudaStat4 == hipSuccess);
    assert(cudaStat5 == hipSuccess);

    //初始化h_Smatrix
    for(int i = 0; i < n*n; ++i){
        h_Smatrix[i] = 0;
    }
  
    for(int i = 0; i < n; ++i){
        h_Smatrix[n*i+i] = h_S[i];
    }

    //裁剪u矩阵
    for(int i = 0; i < m*n; ++i){
        h_Umatrix[i] = h_U[i];
    }

    cudaStat1 = hipMemcpyAsync(d_Umatrix, h_Umatrix, sizeof(dt)*m*n, hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpyAsync(d_Smatrix, h_Smatrix, sizeof(dt)*n*n, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    assert(cudaStat1 == hipSuccess);
    assert(cudaStat2 == hipSuccess);

    GPUTimer timer2;
    timer2.start();
    hipblasStatus_t  statusCublas = hipblasGemmEx(cublasH, 
        HIPBLAS_OP_N, HIPBLAS_OP_T,
        n, m, n,
        &alpha,
        d_Smatrix, HIP_R_32F, n,
        d_Umatrix, HIP_R_32F, m,
        &beta,
        d_AT2, HIP_R_32F, n,
        HIP_R_32F, HIPBLAS_GEMM_DEFAULT);
    assert(statusCublas == HIPBLAS_STATUS_SUCCESS);
    hipDeviceSynchronize();
    time = timer2.seconds();
    cout << "----------------------------------\n part two gemm run time: " << time << endl;

    if(h_U) hipHostFree(h_U);
    if(h_S) hipHostFree(h_S);
    if(h_Umatrix) hipHostFree(h_Umatrix);
    if(h_Smatrix) hipHostFree(h_Smatrix);
    if(d_Smatrix) hipFree(d_Smatrix);
    if(d_Umatrix) hipFree(d_Umatrix);

///计算第三部分svd

    dt *d_A2= NULL;
    dt *d_S2 = NULL;  /* singular values */
    dt *d_U2 = NULL;  /* left singular vectors */
    dt *d_VT2 = NULL;  /* right singular vectors */
    dt *h_G2T = NULL;
    dt *h_G2 = NULL;
    dt *h_G3 = NULL;
    int *devInfo2 = NULL;
    dt *d_work2 = NULL;
    dt *d_rwork2 = NULL;
    
    cudaStat1 = hipMalloc ((void**)&d_A2, sizeof(dt)*m*n);
    assert(cudaStat1 == hipSuccess);
    m = h_n_array[0]*h_n_array[1];
    n = h_n_array[2];

    if(m < n){
        int temp = m;
        m = n;
        n = temp;  // (m > n)
        cublas_status = hipblasSgeam(cublasH,
                                HIPBLAS_OP_T, HIPBLAS_OP_N, 
                                m, n,
                                &alpha,
                                d_AT2, n,
                                &beta,
                                d_A2, m,
                                d_A2, m);
    }
    else{
        cublas_status = hipblasSgeam(cublasH,
                                HIPBLAS_OP_N, HIPBLAS_OP_N, 
                                m, n,
                                &alpha,
                                d_AT2, m,
                                &beta,
                                d_A2, m,
                                d_A2, m);
    }
    assert(cublas_status == HIPSOLVER_STATUS_SUCCESS);

    if (d_AT2) hipFree(d_AT2);
    hipDeviceSynchronize();
    // printMatrix_Device(m,n,d_A2,m,"A2");
    lda = m;  

/* step 3: copy A and B to device */
    
    cudaStat2 = hipMalloc ((void**)&d_S2   , sizeof(dt)*n);
    cudaStat3 = hipMalloc ((void**)&d_U2   , sizeof(dt)*m*m);
    cudaStat4 = hipMalloc ((void**)&d_VT2   , sizeof(dt)*n*n);
    cudaStat5 = hipMalloc ((void**)&devInfo2, sizeof(int));
   
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat4);
    assert(hipSuccess == cudaStat5);
    assert(hipSuccess == cudaStat1);

    cudaStat2 = hipHostAlloc((void**)&h_G2T , sizeof(dt)*m*m, 0);
    cudaStat3 = hipHostAlloc((void**)&h_G2  , sizeof(dt)*m*n, 0);
    cudaStat4 = hipHostAlloc((void**)&h_G3  , sizeof(dt)*n*n, 0);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat4);

// step 3: query working space of SVD
    cusolver_status = hipsolverDnSgesvd_bufferSize(
        cusolverH,
        m,
        n,
        &lwork );
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    cout << "part three work space: " <<sizeof(dt)*lwork << endl;
    cudaStat1 = hipMalloc((void**)&d_work2 , sizeof(dt)*lwork);
    assert(hipSuccess == cudaStat1);

    time = 0.0f;
    GPUTimer timer3;
    timer3.start();
    /* step 5: compute SVD */
    // signed char jobu = 'A'; // all m columns of U
    // signed char jobvt = 'A'; // all n columns of VT
    cusolver_status = hipsolverDnSgesvd(
        cusolverH,
        jobu,
        jobvt,
        m,
        n,
        d_A2,
        lda,
        d_S2,
        d_U2,
        lda,  // ldu
        d_VT2,
        n, // ldvt,
        d_work2,
        lwork,
        d_rwork2,
        devInfo2);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    assert(hipSuccess == cudaStat1);

    time = timer3.seconds();
    cout << "----------------------------------\npart three svd run time: " << time << endl;

    // printMatrix_Device(m,m,d_U2, m,"d_U2");
    // printMatrix_Device(n,1,d_S2, n,"d_S2");
    // printMatrix_Device(n,n,d_VT2, n,"d_VT2");

    cudaStat4 = hipMemcpyAsync(&info_gpu, devInfo2, sizeof(int), hipMemcpyDeviceToHost);
    cudaStat3 = hipMemcpyAsync(h_G2T, d_U2, sizeof(dt)*m*m, hipMemcpyDeviceToHost);
    cudaStat2 = hipMemcpyAsync(h_G3, d_VT2, sizeof(dt)*n*n, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    assert(hipSuccess == cudaStat4);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat2);

    printf("after gesvd: info_gpu = %d\n", info_gpu);
    assert(0 == info_gpu);
    // printf("=====\n");

    for(int i = 0; i < m*n; ++i){
        h_G2[i] = h_G2T[i];
    }

    if (d_A2) hipFree(d_A2);
    if (d_S2) hipFree(d_S2);
    if (d_U2) hipFree(d_U2);
    if (d_VT2) hipFree(d_VT2);
    if (devInfo2) hipFree(devInfo2);
    if (d_work2 ) hipFree(d_work2);
    if (d_rwork2) hipFree(d_rwork2);

    //printf("step2 - calculate mid success--------2*12--------- \n");
    // if(d_AT2) hipHostFree(d_AT2);
    if (h_G2T) hipHostFree(h_G2T);
    if (h_G1) hipHostFree(h_G1);
    if (h_G2) hipHostFree(h_G2);
    if (h_G3) hipHostFree(h_G3);
    if (cublasH ) hipblasDestroy(cublasH);
    if (cusolverH) hipsolverDnDestroy(cusolverH);
    // hipDeviceReset();
    // printf("====finish====");
}