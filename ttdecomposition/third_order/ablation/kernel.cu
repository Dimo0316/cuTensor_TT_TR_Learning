#include "hip/hip_runtime.h"
/*************************************************************************
	> File Name: func.h
	> Author: honghao
	> Mail: honghao@shu.edu.cn 
	> Created Time: 2020年04月2日 星期四 14时25分17秒
 ************************************************************************/
#include "head.h"

__global__  void floattohalf(dt *AA,half *BB,long m){
    long i = blockIdx.x*blockDim.x+threadIdx.x;
    const long temp = blockDim.x*gridDim.x;
    if(i<m){
        BB[i] = __float2half(AA[i]);
        i+=temp;
    }
    __syncthreads();
}


__global__ void matrixGetSub_U(dt *d_src, dt *d_dst, int m, int ks, int k){
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    const long temp = blockDim.x*gridDim.x;
    if(i<m*k){
        d_dst[i]=d_src[i];
        // d_Ainv[i] = d_A[];
        i+=temp;
    }
    __syncthreads();
}

__global__ void matrixGetSub_VT(dt *d_src, dt *d_dst, long ks, long n, long k){
    long i = blockIdx.x*blockDim.x+threadIdx.x;
    const long temp = blockDim.x*gridDim.x;
    if(i<k*n){
        int row = i%k;
        int col = i/k;
        d_dst[i]=d_src[row+col*ks];
        i+=temp;
    }
    __syncthreads();
}

void matrixGetSub_VT_host(dt *d_src, dt *d_dst, long ks, long n, long k){
    dim3 threads(1024,1,1);
    dim3 blocks((n*k+1024-1)/1024,1,1);   
    matrixGetSub_VT<<<blocks,threads>>>(d_src,d_dst,ks,n,k);
}


__global__ void matrixInvertColumn(dt *d_A, dt *d_Ainv, int m, int n){
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    if(i<m*n){
        int row = i%m;
        int col = i/m;
        d_Ainv[row+(n-col-1)*m]=d_A[i];
        // d_Ainv[i] = d_A[];
    }
    __syncthreads();
}


void f2h(dt *A,half *B,long num){
    dim3 threads(1024,1,1);
    dim3 blocks((num+1024-1)/1024,1,1);   
    floattohalf<<<blocks,threads>>>(A,B,num);
}

void genTTTensor(dt *T,long a,long b,long c,long r){
    dt *AA,*BB,*CC;
    hipHostAlloc((void**)&AA,sizeof(dt)*a*r,0);
    hipHostAlloc((void**)&BB,sizeof(dt)*b*r,0);
    hipHostAlloc((void**)&CC,sizeof(dt)*c*r,0);
    for(long i = 0;i<a*r;i++){
        AA[i]=rand()*0.1/(RAND_MAX*0.1)*2.0 - 1.0;
    }
    for(long i = 0;i<b*r;i++){
        BB[i]=rand()*0.1/(RAND_MAX*0.1)*2.0 - 1.0;
    }
    for(long i = 0;i<c*r;i++){
        CC[i]=rand()*0.1/(RAND_MAX*0.1)*2.0 - 1.0;
    }
    dt *d_T,*d_CC,*d_BB,*d_AA;
    hipMalloc((void**)&d_AA,sizeof(dt)*a*r);
    hipMalloc((void**)&d_BB,sizeof(dt)*b*r);
    hipMalloc((void**)&d_CC,sizeof(dt)*c*r);
    hipMalloc((void**)&d_T,sizeof(dt)*a*b*c);
    hipMemcpyAsync(d_BB,BB,sizeof(dt)*b*r,hipMemcpyHostToDevice,0);
    hipMemcpyAsync(d_CC,CC,sizeof(dt)*c*r,hipMemcpyHostToDevice,0);
    hipMemcpyAsync(d_AA,AA,sizeof(dt)*a*r,hipMemcpyHostToDevice,0);
    dt *d_CKRB;
    hipMalloc((void**)&d_CKRB,sizeof(dt)*c*r*b);
    hipDeviceSynchronize();

    // printMatrix_Device(a,r,d_AA,a,"AA");
    // printMatrix_Device(b,r,d_BB,b,"BB");
    // printMatrix_Device(c,r,d_CC,c,"CC");

    //X1=A*(CkrB)'  a*r  r*(bc)
    dt alpha = 1.0;
    dt beta = 0.0;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,
    		b,c,1,
    		&alpha,
    		d_BB,b,b,
    		d_CC,c,c,
    		&beta,d_CKRB,
    		b,b*c,r);
    // printMatrix_Device(b,c*r,d_CKRB,b,"CkrB");

    hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a,b*c,r,&alpha,d_AA,a,d_CKRB,b*c,&beta,d_T,a);
    // printMatrix_Device(a,b*c,d_T,a,"T");

    hipMemcpyAsync(T,d_T,sizeof(dt)*a*b*c,hipMemcpyDeviceToHost,0);
    hipDeviceSynchronize();

    hipFree(d_AA);
    hipFree(d_BB);
    hipFree(d_CC);
    hipFree(d_CKRB);
    hipFree(d_T);
    hipHostFree(AA);
    hipHostFree(BB);
    hipHostFree(CC);
    hipblasDestroy(handle);
}

__global__ void warmup()
{
    int sum=0;
    for(int i = 0; i < 1000; i++) {
        sum+=i;
    }
}   

void warmupcu(){
    warmup<<<1,1>>>();
}

void printMatrix(int m, int n, const dt*A, int lda, const char* name)
{
    for(int row = 0 ; row < m ; row++){
        for(int col = 0 ; col < n ; col++){
            float Areg = A[row + col*lda];
            printf("%s(%d,%d) = %f\n", name, row+1, col+1, Areg);
        }
    }
    cout << " ------------------------------------" << endl;
}

void printMatrix_Device(int m, int n, const dt*d_A, int lda, const char* name)
{
    dt *h_A = NULL;
    hipHostAlloc((void**)&h_A , sizeof(dt)*m*n, 1);
    hipMemcpy(h_A, d_A, sizeof(dt)*m*n, hipMemcpyDeviceToHost);
    printMatrix(m, n, h_A, lda, name);
    if(h_A) hipHostFree(h_A);
}

__global__ void sub(dt *A,dt *B,long a,long b,long c)
{
    long long i = blockIdx.x*blockDim.x+threadIdx.x;
    const long long temp = blockDim.x*gridDim.x;
    while(i<a*b*c)
    {
        B[i] = A[i] - B[i];
        i+=temp;
    }
    __syncthreads();
}

