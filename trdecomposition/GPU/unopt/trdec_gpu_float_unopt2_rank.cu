#include "hip/hip_runtime.h"
#include "head.h"

__global__ void tranforArrayToDiagonalMatrix(float* array, float* diagMatrix, int n){
    long long i = blockIdx.x * blockDim.x + threadIdx.x;
    const long long temp = blockDim.x*gridDim.x;
    while(i < n*n){
        long row = i/n;
        long col = i%n;
        if(row == col)
            diagMatrix[i] = array[row];
        else
            diagMatrix[i] = 0;
        i+=temp;
    }
    __syncthreads();
}

__global__ void hardm(dt *M,dt *N,dt *res,long  m){
     long i = blockIdx.x*blockDim.x+threadIdx.x;
    if(i<m){
        res[m-i] = M[i]*N[i];
    }
    __syncthreads();
}

__global__ void tensorToMode2(dt *T1,dt *T2,int m,int n,int k){
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    if(i<m*n*k){
        int tube = i/(m*n);
        int row = (i-tube*(m*n))%m;
        int col = (i-tube*(m*n))/m;
        T2[tube*m*n+row*n+col] = T1[tube*m*n+col*m+row];
    }
    __syncthreads();
}

__global__ void tensorToMode3(dt *T1,dt *T2,int m,int n,int k){
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    if(i<m*n*k){
        int tube = i/(m*n);
        int row = (i-tube*(m*n))%m;
        int col = (i-tube*(m*n))/m;
        T2[k*(col*m+row)+tube] = T1[tube*m*n+col*m+row];
    }
    __syncthreads();
}

float trdec_gpu_float_unopt(float* h_tensor, int* h_n_array, int* h_r_array, double* eps){
// /计算第一部分svd
    dt time0 = 0.0f;
    GPUTimer timer0;
    timer0.start();

    hipsolverHandle_t cusolverH = NULL;
    hipblasHandle_t cublasH = NULL;
    hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
    hipsolverStatus_t cusolver_status  = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    hipError_t cudaStat4 = hipSuccess;
    hipError_t cudaStat5 = hipSuccess;
    hipError_t cudaStat6 = hipSuccess;


    cusolver_status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

    cublas_status = hipblasCreate(&cublasH);
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

    hipblasSetMathMode(cublasH,HIPBLAS_DEFAULT_MATH);

    dt *d_A = NULL;  /* device copy of A */
    dt *d_AT= NULL;
    dt *d_S = NULL;  /* singular values */
    dt *d_U = NULL;  /* left singular vectors */
    dt *d_VT = NULL;  /* right singular vectors */
    dt *d_G1 = NULL;
    dt *h_G1 = NULL;
    int *devInfo = NULL;
    dt *d_work = NULL;
    dt *d_rwork = NULL;

    int m = h_n_array[0];
    int n = h_n_array[1] * h_n_array[2];
    int lwork = 0;
    int info_gpu = 0;
    float norm_d_S = 0.0f;
    float truncated_Value = 0.0f;
    const dt alpha = 1.0;
    const float ep = eps[0]/1.414214;
    const dt beta = 0.0;

    cudaStat1 = hipMalloc ((void**)&d_A   , sizeof(dt)*m*n);
    cudaStat2 = hipMalloc ((void**)&d_AT  , sizeof(dt)*m*n);
    cudaStat3 = hipMemcpy (d_AT, h_tensor, sizeof(dt)*m*n, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);

    // cout << " m1:" << m << "  n1:" << n << endl;
    
    //printMatrix_Device(m, n, d_AT, m, "A的转置前");
///判断是否要转置
    if(m < n){
        int temp = m;
        m = n;
        n = temp;  // (m > n)
        cublas_status = hipblasSgeam(cublasH,
                                HIPBLAS_OP_T, HIPBLAS_OP_N, 
                                m, n,
                                &alpha,
                                d_AT, n,
                                &beta,
                                d_A, m,
                                d_A, m);
    }
    else{
        cublas_status = hipblasSgeam(cublasH,
                                HIPBLAS_OP_N, HIPBLAS_OP_N, 
                                m, n,
                                &alpha,
                                d_AT, m,
                                &beta,
                                d_AT, n,
                                d_A, m);
    }
    assert(cublas_status == HIPSOLVER_STATUS_SUCCESS);
    if (d_AT) hipFree(d_AT);
    hipDeviceSynchronize();

    int lda = m;  

/* step 3: copy A and B to device */
    
    cudaStat2 = hipMalloc ((void**)&d_S   , sizeof(dt)*n);
    cudaStat3 = hipMalloc ((void**)&d_U   , sizeof(dt)*m*n); //有问题
    cudaStat4 = hipMalloc ((void**)&d_VT   , sizeof(dt)*n*n);
    cudaStat5 = hipMalloc ((void**)&devInfo, sizeof(int));
    cudaStat1 = hipMalloc ((void**)&d_G1  , sizeof(dt)*n*n);

    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);//有问题
    assert(hipSuccess == cudaStat4);
    assert(hipSuccess == cudaStat5);
    assert(hipSuccess == cudaStat1);

    // cudaStat1 = hipHostAlloc((void**)&h_S   , sizeof(dt)*n  , 0);
    
    // cudaStat3 = hipHostAlloc((void**)&h_U   , sizeof(dt)*m*n, 0);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);

    // cout<< " m:" << m << " n:" << n  << " lda:" << lda <<endl;

// step 3: query working space of SVD
    cusolver_status = hipsolverDnSgesvd_bufferSize(
        cusolverH,
        m,
        n,
        &lwork );
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    // cout << "part one work space: " <<sizeof(dt)*lwork << endl;
    cudaStat1 = hipMalloc((void**)&d_work , sizeof(dt)*lwork);
    assert(hipSuccess == cudaStat1);
    // printMatrix_Device(m, n, d_A, m, "A");
    
    /* step 5: compute SVD */
    signed char jobu = 'S'; // all m columns of U
    signed char jobvt = 'S'; // all n columns of VT
    // cout << "m:" << m <<" n:" <<n<< " lda:" <<lda<<endl;
    cusolver_status = hipsolverDnSgesvd(
        cusolverH,
        jobu,
        jobvt,
        m,
        n,
        d_A,
        m,
        d_S,
        d_U,
        m,  // ldu
        d_VT,
        n, // ldvt,
        d_work,
        lwork,
        d_rwork,
        devInfo);
    cudaStat1 = hipDeviceSynchronize();

    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    assert(hipSuccess == cudaStat1);

    // cout << "m:" << m << " n:" << n <<" lda:" << lda <<endl;
    
    // printMatrix_Device(n, 1, d_S, n, "S");
    // printMatrix_Device(m, n, d_U, m, "U");
    // printMatrix_Device(n, n, d_VT, n, "VT");

    // printMatrix_Device(n, n, d_G1, n, "G1");
    // printMatrix_Device(n,1,d_S, n,"d_S");
    cudaStat4 = hipMemcpyAsync(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    assert(hipSuccess == cudaStat4);
    // printf("after gesvd: info_gpu = %d\n", info_gpu);
    assert(0 == info_gpu);
   
    if (d_A) hipFree(d_A);
    if (devInfo) hipFree(devInfo);
    if (d_work ) hipFree(d_work);
    if (d_rwork) hipFree(d_rwork);

    dim3 threads1024(1024,1,1);
    dim3 blocks1024((n+1024-1)/1024,1,1);

//计算二范数。获得截断值
//     dt *d_Shardm = NULL;
//     cudaStat1 = hipMalloc((void**)&d_Shardm, sizeof(dt)*n);
//     assert(cudaStat1 == hipSuccess);
//     cublas_status = hipblasSnrm2(cublasH, n,
//                 d_S, 1, &norm_d_S);
//     assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
//     cudaStat1 = hipDeviceSynchronize();
//     assert(cudaStat1 == hipSuccess);
//     truncated_Value = norm_d_S * ep * norm_d_S * ep;
// ///获得截断索引 truncat_number 也就是h_r_array
// ///TODO
//     float sumError = 0.0f;
//     int truncat_number_test = 0;
//     hardm<<<blocks1024,threads1024>>>(d_S,d_S,d_Shardm,n);
//     // printMatrix_Device(econ_s,1,d_S,econ_s,"S");
//     // printMatrix_Device(econ_s,1,d_Shardm,econ_s,"Shardm");
//     cout << "truncated_Value: " << truncated_Value << endl;
//     while(sumError < truncated_Value){
//         cublas_status = hipblasSasum(cublasH, truncat_number_test,
//                             d_Shardm, 1, &sumError);
//         assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
//         truncat_number_test++;
//     }

//     if(d_Shardm) hipFree(d_Shardm);   

    int truncat_number = h_r_array[0]*h_r_array[1];
    
    // cout << "truncat_number:" << truncat_number << endl;

///tr分解中获得tr里的   r1 和 r2  
	int r1 = h_r_array[0];
	int r2 = h_r_array[1];
	cout << "r1:" << r1 << " r2:" << r2 << endl;
	
///复制G1到内存 h_G1
    cublas_status = hipblasSgeam(cublasH,
                                HIPBLAS_OP_T, HIPBLAS_OP_N, 
                                n, n,
                                &alpha,
                                d_VT, n,
                                &beta,
                                d_VT, n,
                                d_G1, n);
    assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
    cudaStat6 = hipDeviceSynchronize();
    assert(cudaStat6 == hipSuccess);
    if (d_VT   ) hipFree(d_VT);
	

    float *d_Struncat = NULL;
///计算中间变量 第二部分
    dt *d_Smatrix = NULL;
    dt *d_AT2 = NULL, *d_G1truncat=NULL, *d_AT2Mid = NULL;
    cudaStat4 = hipHostAlloc((void**)&h_G1  ,    sizeof(dt)*truncat_number*n, 0);
    cudaStat1 = hipMalloc((void**)&d_Struncat,   sizeof(dt)*truncat_number);
    cudaStat5 = hipMalloc((void**)&d_G1truncat,  sizeof(dt)*truncat_number*n);
    cudaStat2 = hipMalloc((void**) &d_Smatrix,	  sizeof(dt)*truncat_number*truncat_number);
    cudaStat3 = hipMalloc((void**) &d_AT2,       sizeof(dt)*truncat_number*m);
    cudaStat3 = hipMalloc((void**) &d_AT2Mid,    sizeof(dt)*truncat_number*m);
    assert(cudaStat1 == hipSuccess);
    assert(cudaStat2 == hipSuccess);
    assert(cudaStat3 == hipSuccess);
    assert(cudaStat4 == hipSuccess);

///TODO 将G1 从n1*r1*r2 转变成 r1*n1*r2

    dim3 threadsnr1r2(1024,1,1);
    dim3 blocksnr1r2((r1*r2*m+1024-1)/1024,1,1);
    
	cublas_status = hipblasScopy(cublasH, truncat_number*n,
                            d_G1, 1,
                            d_G1truncat, 1);
    assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
    cudaStat1 = hipDeviceSynchronize();
    assert(cudaStat1 == hipSuccess);


	tensorToMode2<<<blocksnr1r2,threadsnr1r2>>>(d_G1truncat, d_G1, n, r1, r2);
	cudaStat6 = hipDeviceSynchronize();
	assert(cudaStat6 == hipSuccess);
    cudaStat5 = hipMemcpyAsync(h_G1, d_G1, sizeof(dt)*truncat_number*n, hipMemcpyDeviceToHost);
    cudaStat6 = hipDeviceSynchronize();
    assert(cudaStat5 == hipSuccess);
    assert(cudaStat6 == hipSuccess);

    cublas_status = hipblasScopy(cublasH, truncat_number,
                            d_S, 1,
                            d_Struncat, 1);
    assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
    cudaStat1 = hipDeviceSynchronize();
    assert(cudaStat1 == hipSuccess);
    if(d_S) hipFree(d_S);
    if(d_G1) hipFree(d_G1);
    if(d_G1truncat) hipFree(d_G1truncat);
    
    // printMatrix_Device(n,n,d_Smatrix,n,"Smatrix1");
    dim3 thread(512,1,1);
    dim3 blocks((n*n+1024-1)/1024,1,1);

    tranforArrayToDiagonalMatrix<<<blocks,thread>>>(d_Struncat,d_Smatrix,truncat_number);
    cudaStat1 = hipDeviceSynchronize();
    assert(cudaStat1 == hipSuccess);
    // printMatrix_Device(n,n,d_Smatrix,n,"Smatrix");
    cublas_status = hipblasGemmEx(cublasH, 
        HIPBLAS_OP_N, HIPBLAS_OP_T,
        truncat_number, m, truncat_number,
        &alpha,
        d_Smatrix, HIP_R_32F, truncat_number,
        d_U, HIP_R_32F, m,
        &beta,
        d_AT2Mid, HIP_R_32F, truncat_number,
        HIP_R_32F, HIPBLAS_GEMM_DEFAULT);
    assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
    cudaStat1 = hipDeviceSynchronize();
    assert(cudaStat1 == hipSuccess);

    dim3 threads512(1024,1,1);
    dim3 blocks512((r1*r2*m+1024-1)/1024,1,1);
    // GPUTimer timer2;
    // timer2.start();
///TODO 将d_AT2 从r1*r2*m 转变成 r2*m*r1
	tensorToMode2<<<blocks512,threads512>>>(d_AT2Mid,d_AT2,r1,r2,m);//r1 r2 m
    cudaStat1 = hipDeviceSynchronize();
    assert(cudaStat1 == hipSuccess);
    tensorToMode3<<<blocks512,threads512>>>(d_AT2,d_AT2Mid,r2,r1,m); //r2 r1 m
    cudaStat1 = hipDeviceSynchronize();
    assert(cudaStat1 == hipSuccess);
    tensorToMode2<<<blocks512,threads512>>>(d_AT2Mid,d_AT2,m,r2,r1); // m r2 r1
    cudaStat1 = hipDeviceSynchronize();
    assert(cudaStat1 == hipSuccess);
    // float time = timer2.seconds();
    // cout << "----------------------------------\nchange gemm run time: " << time << endl;
    // printMatrix_Device(n,m,d_AT2,n,"AT2");
    
    if(d_Smatrix) hipFree(d_Smatrix);
    // if(d_S) hipFree(d_S);
    if(d_U) hipFree(d_U);
    if(d_AT2Mid) hipFree(d_AT2Mid);


///TODO 改变 计算的m n数值
///计算第三部分svd
    cusolverH = NULL;
    cusolver_status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

    dt *d_S2 = NULL;  /* singular values */
    dt *d_U2 = NULL;  /* left singular vectors */
    dt *d_VT2 = NULL;  /* right singular vectors */
    dt *d_G3 = NULL;

    dt *h_G2 = NULL;
    dt *h_G3 = NULL;
    int *devInfo2 = NULL;
    dt *d_work2 = NULL;
    dt *d_rwork2 = NULL;
    
    assert(cudaStat1 == hipSuccess);
    m = h_n_array[1]*r2;
    n = h_n_array[2]*r1;
    // cout << "m:" << m << " n:" << n << endl;
   
    cudaStat1 =  hipDeviceSynchronize();
    assert(hipSuccess == cudaStat1);
    // printMatrix_Device(m,n,d_A2,m,"A2");
    lda = m;  

/* step 3: copy A and B to device */
    cudaStat1 = hipMalloc ((void**)&d_G3   , sizeof(dt)*n*n);
    cudaStat2 = hipMalloc ((void**)&d_S2   , sizeof(dt)*n);
    cudaStat3 = hipMalloc ((void**)&d_U2   , sizeof(dt)*m*n);
    cudaStat4 = hipMalloc ((void**)&d_VT2   , sizeof(dt)*n*n);
    cudaStat5 = hipMalloc ((void**)&devInfo2, sizeof(int));
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat4);
    assert(hipSuccess == cudaStat5);

    int lwork2 = 0;
// step 3: query working space of SVD
    cusolver_status = hipsolverDnSgesvd_bufferSize(
        cusolverH,
        m,
        n,
        &lwork2 );
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    // cout << "part three work space: " << sizeof(dt)*lwork2 << endl;
    cudaStat1 = hipMalloc((void**)&d_work2 , sizeof(dt)*lwork2);
    assert(hipSuccess == cudaStat1);
     // cout << "m:" << m << " n:" << n <<" lda:" << lda <<endl;
    
    cusolver_status = hipsolverDnSgesvd(
        cusolverH,
        jobu,
        jobvt,
        m,
        n,
        d_AT2,
        lda,
        d_S2,
        d_U2,
        m,  // ldu
        d_VT2,
        n, // ldvt,
        d_work2,
        lwork2,
        d_rwork2,
        devInfo2);
    cudaStat1 = hipDeviceSynchronize();
   
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    assert(hipSuccess == cudaStat1);
    if(d_AT2) hipFree(d_AT2);
    
    // printMatrix_Device(n, 1, d_S2, n, "S2");
    // printMatrix_Device(m, n, d_U2, m, "U2");
    // printMatrix_Device(n, n, d_VT2, n, "VT2");
    
    
    //计算获得G3 TODO改成非优化版本
    // cublas_status = hipblasSdgmm(cublasH,
    //                             HIPBLAS_SIDE_LEFT,
    //                             n, n,
    //                             d_V2, n,
    //                             d_S2, 1,
    //                             d_G3, n);
    // assert(statusCublas == HIPBLAS_STATUS_SUCCESS);
    // cudaStat1 = hipDeviceSynchronize();
    // assert(hipSuccess == cudaStat1);

//     //计算二范数。获得截断值
//     dt *d_Shardm2 = NULL;
//     cudaStat1 = hipMalloc((void**)&d_Shardm2, sizeof(dt)*n);
//     assert(cudaStat1 == hipSuccess);
//     cublas_status = hipblasSnrm2(cublasH, n,
//                 d_S2, 1, &norm_d_S);
//     assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
//     cudaStat1 = hipDeviceSynchronize();
//     assert(cudaStat1 == hipSuccess);
//     truncated_Value = norm_d_S * ep * norm_d_S * ep;
//      // cout <<"truncated :" << truncated  << " norm_d_S:" << norm_d_S << endl;
// ///获得截断索引 truncat_number 也就是h_r_array
// ///TODO
//     sumError = 0.0f;
//     truncat_number_test = 0;
//     dim3 blocks10242((n+1024-1)/1024,1,1);
//     hardm<<<blocks10242,threads1024>>>(d_S2,d_S2,d_Shardm2,n);
//     // printMatrix_Device(n,1,d_S2,n,"S2");
//     // printMatrix_Device(econ_s,1,d_Shardm,econ_s,"Shardm");
//     cout << "truncated_Value2: " << truncated_Value << endl;
//     while(sumError < truncated_Value){
//         cublas_status = hipblasSasum(cublasH, truncat_number_test,
//                             d_Shardm2, 1, &sumError);
//         assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
//         // cout << "sumError2" << sumError << endl;
//         truncat_number_test++;
//     }
//     // time = timer2.seconds();
//     // cout << "truncat_number_test2:" << econ_s2 - truncat_number_test + 3 <<endl;
//     if(d_Shardm2) hipFree(d_Shardm2);

///获得截断索引 truncat_number 也就是h_r_array
///TODO

    truncat_number = h_r_array[2];
    cout << "truncat_number2: " << truncat_number << endl;
    
    dt *d_Smatrix2 = NULL, *d_Struncat2 = NULL;//, *d_Vtruncat2 = NULL;
    cudaStat1 = hipMalloc((void**)&d_Struncat2, sizeof(dt)*truncat_number);
    cudaStat2 = hipMalloc((void**) &d_Smatrix2, sizeof(dt)*truncat_number*truncat_number);
    cudaStat3 = hipHostAlloc((void**)&h_G2  , sizeof(dt)*m*truncat_number, 0);
    cudaStat4 = hipHostAlloc((void**)&h_G3  , sizeof(dt)*n*truncat_number, 0);
    cudaStat5 = hipDeviceSynchronize();
    assert(cudaStat1 == hipSuccess);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    assert(cudaStat4 == hipSuccess);
    assert(cudaStat5 == hipSuccess);


    ///截断S2
    cublas_status = hipblasScopy(cublasH, truncat_number,
                            d_S2, 1,
                            d_Struncat2, 1);
    assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
    cudaStat1 = hipDeviceSynchronize();
    assert(cudaStat1 == hipSuccess);
    if(d_S2) hipFree(d_S2);

    // GPUTimer timer4; 
    // timer4.start();
    dim3 thread2(512,1,1);
    dim3 blocks2((n*n+1024-1)/1024,1,1);
    tranforArrayToDiagonalMatrix<<<blocks2,thread2>>>(d_Struncat2,d_Smatrix2,truncat_number);
    cudaStat1 = hipDeviceSynchronize();
    assert(cudaStat1 == hipSuccess);
    if(d_Struncat2) hipFree(d_Struncat2); d_Struncat2=NULL;
    cublas_status = hipblasGemmEx(cublasH, 
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        truncat_number, n, truncat_number,
        &alpha,
        d_Smatrix2, HIP_R_32F, truncat_number,
        d_VT2, HIP_R_32F, truncat_number,
        &beta,
        d_G3, HIP_R_32F, truncat_number,
        HIP_R_32F, HIPBLAS_GEMM_DEFAULT);
    assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
    cudaStat1 = hipDeviceSynchronize();
    assert(cudaStat1 == hipSuccess);

    // printMatrix_Device(n,n,d_G3,n,"G3");

    cudaStat4 = hipMemcpyAsync(&info_gpu, devInfo2, sizeof(int), hipMemcpyDeviceToHost);
    cudaStat3 = hipMemcpyAsync(h_G2, d_U2, sizeof(dt)*m*truncat_number, hipMemcpyDeviceToHost);
    cudaStat2 = hipMemcpyAsync(h_G3, d_G3, sizeof(dt)*n*truncat_number, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    assert(hipSuccess == cudaStat4);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat2);

    // printf("after gesvd: info_gpu = %d\n", info_gpu);
    assert(0 == info_gpu);
  
    // if (d_S2) hipFree(d_S2);
    if (d_U2) hipFree(d_U2);
    if (d_VT2) hipFree(d_VT2);
    if (d_G3) hipFree(d_G3);
    if (devInfo2) hipFree(devInfo2);
    if (d_work2 ) hipFree(d_work2);
    if (d_rwork2) hipFree(d_rwork2);
    if (d_Smatrix2) hipFree(d_Smatrix2);
    
    if (h_G1) hipHostFree(h_G1);
    if (h_G2) hipHostFree(h_G2);
    if (h_G3) hipHostFree(h_G3);
    if (cublasH ) hipblasDestroy(cublasH);
    if (cusolverH) hipsolverDnDestroy(cusolverH);

    time0 = timer0.seconds();
    return time0;
}