#include "hip/hip_runtime.h"
/*************************************************************************
	> File Name: func.h
	> Author: honghao
	> Mail: honghao@shu.edu.cn 
	> Created Time: 2020年04月2日 星期四 14时25分17秒
 ************************************************************************/
#include "head.h"

__global__ void warmup()
{
    int sum=0;
    for(int i = 0; i < 1000; i++) {
        sum+=i;
    }
}   

void warmupcu(){
    warmup<<<1,1>>>();
}


void genTRTensor(dt *T,long a,long b,long c,long r){
    dt *AA,*BB,*CC;    
    hipHostAlloc((void**)&AA,sizeof(dt)*a*r,0);
    hipHostAlloc((void**)&BB,sizeof(dt)*b*r,0);
    hipHostAlloc((void**)&CC,sizeof(dt)*c*r,0);
    for(long i = 0;i<a*r;i++){
        AA[i]=rand()*0.1/(RAND_MAX*0.1);
    }
    for(long i = 0;i<b*r;i++){
        BB[i]=rand()*0.1/(RAND_MAX*0.1);
    }
    for(long i = 0;i<c*r;i++){
        CC[i]=rand()*0.1/(RAND_MAX*0.1);
    }
    dt *d_T,*d_CC,*d_BB,*d_AA;
    hipMalloc((void**)&d_AA,sizeof(dt)*a*r);
    hipMalloc((void**)&d_BB,sizeof(dt)*b*r);
    hipMalloc((void**)&d_CC,sizeof(dt)*c*r);
    hipMalloc((void**)&d_T,sizeof(dt)*a*b*c);
    hipMemcpyAsync(d_BB,BB,sizeof(dt)*b*r,hipMemcpyHostToDevice,0);
    hipMemcpyAsync(d_CC,CC,sizeof(dt)*c*r,hipMemcpyHostToDevice,0);
    hipMemcpyAsync(d_AA,AA,sizeof(dt)*a*r,hipMemcpyHostToDevice,0);
    dt *d_CKRB;
    hipMalloc((void**)&d_CKRB,sizeof(dt)*c*r*b);
    hipDeviceSynchronize();

    //X1=A*(CkrB)'  a*r  r*(bc)
    dt alpha = 1.0;
    dt beta = 0.0;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,
            b,c,1,
            &alpha,
            d_BB,b,b,
            d_CC,c,c,
            &beta,d_CKRB,
            b,b*c,r);

    hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a,b*c,r,&alpha,d_AA,a,d_CKRB,b*c,&beta,d_T,a);

    hipMemcpyAsync(T,d_T,sizeof(dt)*a*b*c,hipMemcpyDeviceToHost,0);
    hipDeviceSynchronize();

    hipFree(d_AA);
    hipFree(d_BB);
    hipFree(d_CC);
    hipFree(d_CKRB);
    hipFree(d_T);
    hipHostFree(AA);
    hipHostFree(BB);
    hipHostFree(CC);
    hipblasDestroy(handle);
}

void printMatrix(int m, int n, const dt*A, int lda, const char* name)
{
    for(int row = 0 ; row < m ; row++){
        for(int col = 0 ; col < n ; col++){
            float Areg = A[row + col*lda];
            printf("%s(%d,%d) = %f\n", name, row+1, col+1, Areg);
        }
    }
    cout << " ------------------------------------" << endl;
}

void printMatrix_Device(int m, int n, const dt*d_A, int lda, const char* name)
{
    dt *h_A = NULL;
    hipHostAlloc((void**)&h_A , sizeof(dt)*m*n, 1);
    hipMemcpy(h_A, d_A, sizeof(dt)*m*n, hipMemcpyDeviceToHost);
    printMatrix(m, n, h_A, lda, name);
    if(h_A) hipHostFree(h_A);
}

int getMinR1R2(int n){
    int a =1, b = n;
    for(int i = 1; i*i<=100;++i){
        if(n%i==0){
            if(n/i-i < b-a){
                a=i;b=n/i;
            }
        }
    }
    return a;
}