// #pragma once
#include "layers.h"

void layerFull::initLayer(int num1,int num2,int batch)
{
    dt *host_W = new dt[num1*num2];
    hipMalloc((void**)&value,sizeof(dt)*num1*batch);
    hipMalloc((void**)&W,sizeof(dt)*num1*num2);
    srand((unsigned)time(NULL));
    for (size_t i = 0; i < num1*num2; i++)//给权值赋一个随机值
    {
        host_W[i] = rand() % 100 / dt(100)*0.1;
        if (rand() % 2)
        {
            host_W[i] = -host_W[i];
        }
    }
    hipMemcpy(W,host_W,sizeof(dt)*num1*num2,hipMemcpyHostToDevice);
    hipDeviceSynchronize();
}

void layerFull::malloc_W(int num1,int num2,int batch)
{
    hipMalloc((void**)&value,sizeof(dt)*num1*batch);
    hipMalloc((void**)&W,sizeof(dt)*num1*num2);
}

layerFull::~layerFull(){
    // printf("This is invoke of layerFull\n");
    if(value) hipFree(value); value = NULL;
    if(W) hipFree(W); W = NULL;
}

void layerTTG::initLayer(int out_mode, int rank2, int rank1, int inp_mode, int out_value2, int batch)
{
    dt *host_W = NULL;
    int number_W = out_mode*rank2*rank1*inp_mode;
    int number_Value = out_mode*rank2*out_value2*batch;
    hipHostAlloc((void**)&host_W,sizeof(dt)*number_W,0);
    hipMalloc((void**)&value,sizeof(dt)*number_Value);
    hipMalloc((void**)&G,sizeof(dt)*number_W);
    srand((unsigned)time(NULL));
    for (size_t i = 0; i < number_W; i++)//给权值赋一个随机值
    {
        host_W[i] = rand() % 100 / dt(100)*0.1;
        if (rand() % 2)
        {
            host_W[i] = -host_W[i];
        }
    }
    hipMemcpy(G,host_W,sizeof(dt)*number_W,hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    if(host_W) hipHostFree(host_W); host_W = NULL;
}

void layerTTG::malloc_G(int out_mode, int rank2, int rank1, int inp_mode, int out_value2, int batch)
{
    hipMalloc((void**)&value,sizeof(dt)*out_mode*rank2*out_value2*batch);
    hipMalloc((void**)&G,sizeof(dt)*out_mode*rank2*rank1*inp_mode);
}

layerTTG::~layerTTG(){
    // printf("This is invoke of layerTTG\n");
    if(value) hipFree(value); value = NULL;
    if(G) hipFree(G); G = NULL;
}