// #pragma once
#include "net.h"

net::net() {
    //初始化输入与隐藏偏置权重
    srand((unsigned)time(NULL));
    // b1 = 0.0;
    // b2 = 0.0;
    b2 = (rand() % 100) / (dt)100;
    b1= (rand() % 100) / (dt)100;
    rate = 0.025; //学习率
    O = new dt[OPNNUM*BATCHSIZE];
    //初始化输入到隐藏节点个数
    //init matrix W1 IPNNUM*HDNNUM
   
    // inlayer.init_W(IPNNUM,K2,BATCHSIZE);
    // hidlayer_U2.init_W(K2,K1,BATCHSIZE);
    // hidlayer_B1.init_W(K1,HDNNUM,BATCHSIZE);
    // inlayer_tmp.initLayer(IPNNUM,HDNNUM,BATCHSIZE); 
    // hipblasCreate(&handle);
    // ht(inlayer_tmp.W,inlayer.W,hidlayer_B1.W,hidlayer_U2.W,HDNNUM,IPNNUM,K1,K2,handle);
    // hidlayer_U1.initLayer(HDNNUM,OPNNUM,BATCHSIZE);
    // //hidlayer.initLayer(HDNNUM,OPNNUM,BATCHSIZE);
    // outlayer.initLayer(OPNNUM,OPNNUM,BATCHSIZE);
    // hipMalloc((void**)&Tg,sizeof(dt)*OPNNUM*BATCHSIZE);
    out_values1[0] = IPNNUM/(ranks_1[0]*inp_modes_1[0]);
    out_values1[1] = out_values1[0]*out_modes_1[0]/(inp_modes_1[1]);
    out_values1[2] = out_values1[1]*out_modes_1[1]/(inp_modes_1[2]);
    out_values1[3] = out_values1[2]*out_modes_1[2]/(inp_modes_1[3]);
    
    // for(int i=0;i<4;i++){
    //     printf("%d\n", out_values1[i]);
    // }
    inlayer.initLayer(IPNNUM,IPNNUM,BATCHSIZE);
    hidlayer_TT1_G1.initLayer(out_modes_1[0], ranks_1[1], ranks_1[0], inp_modes_1[0], out_values1[0], BATCHSIZE);
    hidlayer_TT1_G2.initLayer(out_modes_1[1], ranks_1[2], ranks_1[1], inp_modes_1[1], out_values1[1], BATCHSIZE);
    hidlayer_TT1_G3.initLayer(out_modes_1[2], ranks_1[3], ranks_1[2], inp_modes_1[2], out_values1[2], BATCHSIZE);
    hidlayer_TT1_G4.initLayer(out_modes_1[3], ranks_1[4], ranks_1[3], inp_modes_1[3], out_values1[3], BATCHSIZE);
    outlayer.initLayer(OPNNUM,HDNNUM,BATCHSIZE);
    hipMalloc((void**)&Tg,sizeof(dt)*OPNNUM*BATCHSIZE);
    hipblasCreate(&handle);
///TODO ttDecomposition
}


net::~net(){
	// printf("This is invoke of net\n");
	if(handle) hipblasDestroy(handle); handle = NULL;
	if(Tg) hipFree(Tg); Tg = NULL;
	delete O;
}

//损失函数
dt net::getloss() {
    dt mloss = 0;
    mloss = loss_gpu_cross(Tg,outlayer.value,OPNNUM,BATCHSIZE,handle);  
    return mloss;
}


//前向
void net::forward(dt *input) {
    hipMemcpy(inlayer.value,input,sizeof(dt)*IPNNUM*BATCHSIZE,hipMemcpyHostToDevice);

    forward_cuda(inlayer.value, hidlayer_TT1_G1.G, 
    			 hidlayer_TT1_G1.value, hidlayer_TT1_G2.G,
                 hidlayer_TT1_G2.value, hidlayer_TT1_G3.G,
                 hidlayer_TT1_G3.value, hidlayer_TT1_G4.G,
                 hidlayer_TT1_G4.value, outlayer.W,
                 outlayer.value, b1,b2, 
                 inp_modes_1, out_modes_1, ranks_1, out_values1, OPNNUM, BATCHSIZE, handle);
    
}

//反向
void net::backward(dt *T) {
    hipMemcpy(Tg, T, sizeof(dt)*OPNNUM*BATCHSIZE,hipMemcpyHostToDevice);    
///TODO
    back_cuda(Tg, outlayer.value, outlayer.W, 
    			hidlayer_TT1_G4.value, hidlayer_TT1_G4.G, hidlayer_TT1_G3.value, hidlayer_TT1_G3.G, 
    			hidlayer_TT1_G2.value, hidlayer_TT1_G2.G, hidlayer_TT1_G1.value, hidlayer_TT1_G1.G,
    			inlayer.value,
    			inp_modes_1, out_modes_1, ranks_1, out_values1, OPNNUM, BATCHSIZE, rate, handle);
}

void net::printresual(int trainingTimes)
{
    dt loss = getloss();
    cout << "train times:" << trainingTimes << endl;
    cout << "loss:" << loss << endl;
}

