#include "hip/hip_runtime.h"
#include "head.h"

__global__ void upper1(double *A,double *R,int m,int n)
{
  long long i = blockIdx.x*blockDim.x+threadIdx.x;
  const long long temp = blockDim.x*gridDim.x;
   while(i<n*n)
  { 
    long row=i/n;
    long col=i%n;
    if(row>=col)  
      R[i]=A[row*m+col];
    else
      R[i]=0;
    i+=temp;    
  }
  __syncthreads();
}
void basicQR(hipsolverHandle_t cusolverH, double *d_A, double *d_R, int m, int n){
    // printf("start qr for m: %d, n:%d\n", m,n);
    double *d_work = NULL, *d_tau = NULL;
    int *devInfo = NULL;
    int  lwork = 0; 
    hipMalloc((void**)&d_tau, sizeof(double)*n);
    hipMalloc((void**)&devInfo, sizeof(int));
    hipsolverDnDgeqrf_bufferSize(
        cusolverH, 
        m, 
        n, 
        d_A, 
        m, 
        &lwork);
    hipMalloc((void**)&d_work, sizeof(double)*lwork );
    hipsolverDnDgeqrf(
        cusolverH, 
        m, 
        n, 
        d_A, 
        m, 
        d_tau, 
        d_work, 
        lwork, 
        devInfo);
    hipDeviceSynchronize();
    dim3 blockR((n*n+1024-1)/1024,1,1);
    upper1<<<blockR,1024>>>(d_A,d_R,m,n);

    hipsolverDnDorgqr(
        cusolverH,
        m,
        n,
        n,
        d_A,
        m,
        d_tau,
        d_work,
        lwork,
        devInfo);
    hipDeviceSynchronize();

    if (d_work) hipFree(d_work); d_work = NULL;
    if (devInfo) hipFree(devInfo); devInfo = NULL;
    if (d_tau) hipFree(d_tau); d_tau = NULL;
}

__global__ void Vector2_Multiply_By_Elements (const double* a, const double* b, int n, double* out){
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	const long temp = blockDim.x*gridDim.x;
	while(tid<n)
	{
		out[tid]=a[tid]*(1.0-a[tid])*(b[tid]-a[tid]);
		tid+=temp;
	}
	__syncthreads();
}

__global__ void Vector1_Multiply_By_Elements (const double* a, double* b, int n){
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	const long temp = blockDim.x*gridDim.x;
	while(tid<n)
	{
		b[tid]=a[tid]*(1.0-a[tid])*b[tid];
		tid+=temp;
	}
	__syncthreads();
}

__global__ void activate(double *d_A,int b,int a)
{
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	const long temp = blockDim.x*gridDim.x;
	while(tid<a)
	{
		d_A[tid] = 1/(1+exp(-d_A[tid]+b));
		tid+=temp;
	}
	__syncthreads();
}
void printTensor(double *d_des,long m,long n,long l){
	double *des = new double[m*n*l]();
	hipMemcpy(des,d_des,sizeof(double)*m*n*l,hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	for(long k = 0;k<l;k++){
		for(long i = 0;i<n;i++){
			for(long j = 0;j<m;j++){
				cout<<des[k*m*n+i*m+j]<<" ";
			}
			cout<<endl;
		}
		cout<<"~~~~~~~~~~~~~~~~"<<endl;
	}
	delete[] des;des=nullptr;
}
void warmup(){
	double *tmp = new double[9];
	for(unsigned i = 0; i < 9; ++i) {
		tmp[i] = i+1;
	}
	double *d_tmp;
	hipMalloc((void**)&d_tmp,sizeof(double)*9);
	hipMemcpy(d_tmp,tmp,sizeof(double)*9,hipMemcpyHostToDevice);
	activate<<<1,512>>>(d_tmp,3,3);
	hipDeviceSynchronize();
	hipFree(d_tmp);

}
/*forward_cuda(inlayer.value,inlayer.W,hidlayer_U2.value,hidlayer_U2.W,
                 hidlayer_B1.value,hidlayer_B1.W,hidlayer_U1.value,hidlayer_U1.W,outlayer.value,
                 k1,k2,
                 IPNNUM,HDNNUM,OPNNUM,BATCHSIZE,handle);*/
void forward_cuda(double *input,double *W_U2,double *outu2,double *W_B1,
                  double *outB1,double *W_U1,double *outU1,double *W2,double *outo,
                  double b1,double b2,
                  int in,int k2,int k1,int hid,int out,int batch,hipblasHandle_t handle)
{
	//hid 行 in列 W1 ,out行，hid列 W2

	double alpha=1.0, beta=0.0;
	hipblasDgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,k2,batch,in,&alpha,W_U2,k2,input,in,&beta,outu2,k2);
	hipblasDgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,k1,batch,k2,&alpha,W_B1,k1,outu2,k2,&beta,outB1,k1);
	hipblasDgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,hid,batch,k1,&alpha,W_U1,hid,outB1,k1,&beta,outU1,hid);


	//激活函数
	dim3 blockh((batch*hid+1024-1)/1024,1,1);
	activate<<<blockh,1024>>>(outU1,b1,hid*batch);
	hipDeviceSynchronize();

	dim3 blocko((batch*out+1024-1)/1024,1,1);
	hipblasDgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,out,batch,hid,&alpha,W2,out,outU1,hid,&beta,outo,out);	
	activate<<<blocko,1024>>>(outo,b2,out*batch);
	hipDeviceSynchronize();
	//printTensor(outo,10,1,1);

}
/*
back_cuda(Tg,outlayer.value,
			hidlayer_U1.value,hidlayer_U1.W,
			hidlayer_B1.value,hidlayer_B1.W,
              hidlayer_U2.value,hidlayer_U2.W,
              inlayer.value,inlayer.W,IPNNUM,K2,K1,HDNNUM,OPNNUM,BATCHSIZE,handle
              );*/
void back_cuda(double *Y,double *Y_hat,
               double *outU1,double *W2,
               double *outB1,double *W_U1,
               double *outu2,double *W_B1,
               double *input,double *W_U2,int in,int k2,int k1,int hid,int out,int batch,double rate,hipblasHandle_t handle)
{

	double *d_thta3,*d_thta2,*d_thta1,*d_thta0;
	hipMalloc((void**)&d_thta3,sizeof(double)*out*batch);
	hipMalloc((void**)&d_thta2,sizeof(double)*hid*batch);
	hipMalloc((void**)&d_thta1,sizeof(double)*k1*batch);
	hipMalloc((void**)&d_thta0,sizeof(double)*k2*batch);

	dim3 block2((batch*out+1024-1)/1024,1,1);
	Vector2_Multiply_By_Elements<<<block2,1024>>>(Y_hat, Y, out*batch, d_thta3);
	double alpha=1.0, beta=0.0;
	hipblasDgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,hid,batch,out,&alpha,W2,out,d_thta3,out,&beta,d_thta2,hid);
	
	dim3 block1((batch*hid+1024-1)/1024,1,1);
	Vector1_Multiply_By_Elements<<<block1,1024>>>(outU1, d_thta2, hid*batch);
	hipDeviceSynchronize();

	hipblasDgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,k1,batch,hid,&alpha,W_U1,hid,d_thta2,hid,&beta,d_thta1,k1);
	hipblasDgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,k2,batch,k1,&alpha,W_B1,k1,d_thta1,k1,&beta,d_thta0,k2);

	alpha=rate; beta=1.0;
	hipblasDgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,out,hid,batch,&alpha,d_thta3,out,outU1,hid,&beta,W2,out);
	hipblasDgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,hid,k1,batch,&alpha,d_thta2,hid,outB1,k1,&beta,W_U1,hid);	
	hipblasDgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,k1,k2,batch,&alpha,d_thta1,k1,outu2,k2,&beta,W_B1,k1);
	hipblasDgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,k2,in,batch,&alpha,d_thta0,k2,input,in,&beta,W_U2,k2);
	hipDeviceSynchronize();

	hipFree(d_thta0);
	hipFree(d_thta1);
	hipFree(d_thta2);
	hipFree(d_thta3);
}	

double loss_gpu(double *A,double *B,int n,int batch,hipblasHandle_t handle)
{	
	//A 实际值  B预测值

	double alpha1 = -1.0,loss;
	double *tmp;
	hipMalloc((void**)&tmp,sizeof(double)*n*batch);

	hipblasDcopy(handle,n*batch,B,1,tmp,1);

	hipblasDaxpy(handle,n*batch,&alpha1,A,1,tmp,1);
	hipblasDnrm2(handle,n*batch,tmp,1,&loss); 
	hipDeviceSynchronize();
	hipFree(tmp);
	return loss;
}

void basicSvd(hipsolverHandle_t cusolverH, double *d_B, const int m, const int n, double *d_U, double *d_S, double *d_VT){
    assert(m>n);
    // double alpha = 1.0,beta = 0.0;	
    double *d_work = NULL, *d_rwork = NULL;
    int *devInfo = NULL;
    int lwork = 0;

    hipMalloc((void**)&devInfo, sizeof(int));
	hipsolverDnDgesvd_bufferSize(cusolverH,m,n,&lwork );
	hipMalloc((void**)&d_work , sizeof(double)*lwork);

    hipsolverDnDgesvd(cusolverH,'S','S',m,n,
        d_B,m,d_S,d_U,m,d_VT,n, // ldvt,
        d_work,lwork,d_rwork,devInfo);
    hipDeviceSynchronize();
 
    if(d_work) hipFree(d_work); d_work = NULL;
    if(devInfo) hipFree(devInfo); devInfo = NULL;
    if(d_rwork) hipFree(d_rwork); d_rwork = NULL;
}
void basicSvd_trans(hipsolverHandle_t cusolverH, hipblasHandle_t cublasH, double *d_B, const int m, const int n, double *d_UT, double *d_S, double *d_VT){
    double alpha = 1.0,beta = 0.0;  
    assert(m<n);
    // printf("start svd for m: %d, n:%d\n", m,n);
    double *d_BT = NULL, *d_U = NULL, *d_V=NULL;
    double *d_work = NULL, *d_rwork = NULL;
    int *devInfo = NULL;
    int lwork = 0;

    hipMalloc((void**)&d_BT, sizeof(double)*m*n);
    hipMalloc((void**)&d_U, sizeof(double)*m*m);
    hipMalloc((void**)&d_V, sizeof(double)*m*n);
    hipMalloc ((void**)&devInfo, sizeof(int));
   
//转置B
    hipblasDgeam(cublasH,HIPBLAS_OP_T, HIPBLAS_OP_N, 
                n, m,&alpha,d_B, m,&beta,d_B, n,d_BT, n);
    hipDeviceSynchronize();
    hipsolverDnDgesvd_bufferSize(cusolverH,n,m,&lwork );
    hipMalloc((void**)&d_work , sizeof(double)*lwork);

    signed char jobu = 'S'; // all m columns of U
    signed char jobvt = 'S'; // all n columns of VT
    // cout << "n: " << n <<" m:" << m <<endl;
    hipsolverDnDgesvd(cusolverH,jobu,jobvt,n,m,
        d_BT,n,d_S,d_V,n,d_U,m, // ldvt,
        d_work,lwork,d_rwork,devInfo);
    hipDeviceSynchronize();
    
//转置U 给UT
    hipblasDgeam(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, 
                            m, m,
                            &alpha,
                            d_U, m,
                            &beta,
                            d_U, m,
                            d_UT, m);
    hipDeviceSynchronize();
    hipblasDgeam(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, 
                            m, n,
                            &alpha,
                            d_V, n,
                            &beta,
                            d_V, n,
                            d_VT, m);
    hipDeviceSynchronize();

    if(d_BT) hipFree(d_BT); d_BT = NULL;
    if(d_U) hipFree(d_U); d_U = NULL;
    if(d_work) hipFree(d_work); d_work = NULL;
    if(devInfo) hipFree(devInfo); devInfo = NULL;
    if(d_rwork) hipFree(d_rwork); d_rwork = NULL;
}

void tt(double *d_A,double *G1, double *G2,double *G3, int m, int n, int k1, int k2, hipblasHandle_t handle)
{	
    hipsolverHandle_t cusolverH = NULL;
    hipsolverDnCreate(&cusolverH);
    cout << m << n <<endl;
    double *d_U1=NULL, *d_S1=NULL, *d_VT1=NULL, *d_A2=NULL;
    hipMalloc((void**)&d_U1, sizeof(double)*m*m);
    hipMalloc((void**)&d_S1, sizeof(double)*m);
    hipMalloc((void**)&d_VT1, sizeof(double)*m*n);
    hipMalloc((void**)&d_A2, sizeof(double)*k1*n);

    double *d_S2=NULL, *d_VT2=NULL;
    hipMalloc((void**)&d_S2, sizeof(double)*k2);
    hipMalloc((void**)&d_VT2, sizeof(double)*k2*n);

    basicSvd_trans(cusolverH, handle, d_A, m, n, d_U1, d_S1, d_VT1);

    hipblasDcopy(handle, m*k1, d_U1, 1, G1, 1);
    hipblasDdgmm(handle, HIPBLAS_SIDE_LEFT, k1, n, d_VT1, m, d_S1, 1, d_A2, k1);
   	
    basicSvd_trans(cusolverH, handle, d_A2, k1, n, G2, d_S2, d_VT2);

    hipblasDdgmm(handle, HIPBLAS_SIDE_LEFT, k2, n, d_VT2, k2, d_S2, 1, G3, k2);

    hipFree(d_U1);
    hipFree(d_S1);
    hipFree(d_VT1);
    hipFree(d_A2);

    hipFree(d_S2);
    hipFree(d_VT2);
    // hipFree(d_Gtemp);
    // hipFree(d_Gtemp2);
    hipsolverDnDestroy(cusolverH);
}

__global__ void Gsmaller(double *A,int m)
{
  long long i = blockIdx.x*blockDim.x+threadIdx.x;
  const long long temp = blockDim.x*gridDim.x;
   while(i<m)
  { 
    A[i] = A[i]*0.005;
    i+=temp;    
  }
  __syncthreads();
}

// void tt_random(double *d_A, double *G1, double *G2,double *G3, int m, int n, int k1, int k2, hipblasHandle_t handle)
// {	
// 	hipsolverHandle_t cusolverH = NULL;
//     hipsolverDnCreate(&cusolverH);
//     double alpha = 1.0, beta = 0.0, alpha2 = 0.005;
// 	hiprandGenerator_t gen;
//     hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
//     hiprandSetPseudoRandomGeneratorSeed(gen, 1233456);
//     hiprandGenerateNormalDouble(gen, G1, n*k1, 0, 1); 
//     hiprandGenerateNormalDouble(gen, G2, k1*k2, 0, 1);
//     hiprandGenerateNormalDouble(gen, G3, m*k2, 0, 1);
//     // m = hid      n = input 
//     cout << m << n <<endl;
//     double *d_R1=NULL, *d_R2=NULL;
//     hipMalloc((void**)&d_R1, sizeof(double)*n*k1);
//     hipMalloc((void**)&d_R2, sizeof(double)*k1*k2);

//     basicQR(cusolverH, G1, d_R1, n, k1);
//     hipblasDgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,k1,k1,k1,&alpha,d_R1,k1,G2,k1,&beta,G2,k1);

//     basicQR(cusolverH, G2, d_R2, k1, k1);
//     hipblasDgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,k1,m,k1,&alpha2,d_R2,k1,G3,k1,&beta,G3,k1);

//     dim3 blockG1((n*k1+1024-1)/1024,1,1);
//     Gsmaller<<<blockG1,1024>>>(G1,n*k1);
//     dim3 blockG2((k1*k1+1024-1)/1024,1,1);
//     Gsmaller<<<blockG2,1024>>>(G2,k1*k1);

//     hipFree(d_R1);
//     hipFree(d_R2);
//     hipsolverDnDestroy(cusolverH);
// }